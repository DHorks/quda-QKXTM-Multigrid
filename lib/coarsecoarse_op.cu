#include <transfer.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <complex_quda.h>
#include <index_helper.cuh>
#include <gamma.cuh>
#include <blas_magma.h>
#include <coarse_op.cuh>

namespace quda {

  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, 
            int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, QudaDiracType dirac, QudaMatPCType matpc) {

    typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
    typedef typename colorspinor::FieldOrderCB<Float,2*fineSpin,fineColor,coarseColor,csOrder> F2;
    typedef typename gauge::FieldOrder<Float,fineColor*fineSpin,fineSpin,gOrder> gFine;
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> gCoarse;

    const ColorSpinorField &v = T.Vectors();
    int dummy = 0;
    v.exchangeGhost(QUDA_INVALID_PARITY, dummy);

    F vAccessor(const_cast<ColorSpinorField&>(v));
    F2 uvAccessor(const_cast<ColorSpinorField&>(uv));
    gFine gAccessor(const_cast<GaugeField&>(g));
    gFine cAccessor(const_cast<GaugeField&>(clover));
    gFine cInvAccessor(const_cast<GaugeField&>(cloverInv));
    gCoarse yAccessor(const_cast<GaugeField&>(Y));
    gCoarse xAccessor(const_cast<GaugeField&>(X)); 
    gCoarse xInvAccessor(const_cast<GaugeField&>(Xinv));

    calculateY<true,Float,fineSpin,fineColor,coarseSpin,coarseColor,gOrder>
      (yAccessor, xAccessor, xInvAccessor, uvAccessor, vAccessor, vAccessor, gAccessor, cAccessor, cInvAccessor,
       Y, X, Xinv, Yhat, const_cast<ColorSpinorField&>(v), v, kappa, mu, dirac, matpc);
  }


  // template on the number of coarse degrees of freedom
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int fineColor, int fineSpin>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, QudaDiracType dirac, QudaMatPCType matpc) {
    if (T.Vectors().Nspin()/T.Spin_bs() != 2) 
      errorQuda("Unsupported number of coarse spins %d\n",T.Vectors().Nspin()/T.Spin_bs());
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 2) { 
      calculateYcoarse<Float,csOrder,gOrder,fineColor,fineSpin,2,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else if (coarseColor == 8) {
      calculateYcoarse<Float,csOrder,gOrder,fineColor,fineSpin,8,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else if (coarseColor == 16) {
      calculateYcoarse<Float,csOrder,gOrder,fineColor,fineSpin,16,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else if (coarseColor == 24) {
      calculateYcoarse<Float,csOrder,gOrder,fineColor,fineSpin,24,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else if (coarseColor == 32) {
      calculateYcoarse<Float,csOrder,gOrder,fineColor,fineSpin,32,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // template on fine spin
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int fineColor>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, QudaDiracType dirac, QudaMatPCType matpc) {
    if (T.Vectors().Nspin() == 2) {
      calculateYcoarse<Float,csOrder,gOrder,fineColor,2>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else {
      errorQuda("Unsupported number of spins %d\n", T.Vectors().Nspin());
    }
  }

  // template on fine colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, QudaDiracType dirac, QudaMatPCType matpc) {
    if (g.Ncolor()/T.Vectors().Nspin() == 2) {
      calculateYcoarse<Float,csOrder,gOrder,2>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else if (g.Ncolor()/T.Vectors().Nspin() == 8) {
      calculateYcoarse<Float,csOrder,gOrder,8>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else if (g.Ncolor()/T.Vectors().Nspin() == 16) {
      calculateYcoarse<Float,csOrder,gOrder,16>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else if (g.Ncolor()/T.Vectors().Nspin() == 24) {
      calculateYcoarse<Float,csOrder,gOrder,24>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else if (g.Ncolor()/T.Vectors().Nspin() == 32) {
      calculateYcoarse<Float,csOrder,gOrder,32>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else {
      errorQuda("Unsupported number of colors %d\n", g.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder csOrder>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, QudaDiracType dirac, QudaMatPCType matpc) {
    if (g.FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      calculateYcoarse<Float,csOrder,QUDA_QDP_GAUGE_ORDER>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else {
      errorQuda("Unsupported field order %d\n", g.FieldOrder());
    }
  }

  template <typename Float>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, QudaDiracType dirac, QudaMatPCType matpc) {
    if (T.Vectors().FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      calculateYcoarse<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else {
      errorQuda("Unsupported field order %d\n", T.Vectors().FieldOrder());
    }
  }

  //Does the heavy lifting of creating the coarse color matrices Y
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, ColorSpinorField &uv,
			const Transfer &T, const GaugeField &g, const GaugeField &clover, const GaugeField &cloverInv,
			double kappa, double mu, QudaDiracType dirac, QudaMatPCType matpc) {
    if (X.Precision() != Y.Precision() || Y.Precision() != uv.Precision() || 
        Y.Precision() != T.Vectors().Precision() || Y.Precision() != g.Precision())
      errorQuda("Unsupported precision mix");

    printfQuda("Computing Y field......\n");
    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      calculateYcoarse<double>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      calculateYcoarse<float>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, dirac, matpc);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    printfQuda("....done computing Y field\n");
  }

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void CoarseCoarseOp(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
		      const Transfer &T, const cpuGaugeField &gauge, const cpuGaugeField &clover, const cpuGaugeField &cloverInv,
		      double kappa, double mu, QudaDiracType dirac, QudaMatPCType matpc) {

    QudaPrecision precision = Y.Precision();
    //First make a cpu gauge field from the cuda gauge field

#if 0
    GaugeFieldParam gf_param(gauge.X(), precision, gauge.Reconstruct(), pad, gauge.Geometry());
    gf_param.order = QUDA_QDP_GAUGE_ORDER;
    gf_param.fixed = gauge.GaugeFixed();
    gf_param.link_type = gauge.LinkType();
    gf_param.t_boundary = gauge.TBoundary();
    gf_param.anisotropy = gauge.Anisotropy();
    gf_param.gauge = NULL;
    gf_param.create = QUDA_NULL_FIELD_CREATE;
    gf_param.siteSubset = QUDA_FULL_SITE_SUBSET;

    cpuGaugeField g(gf_param);

    //Copy the cuda gauge field to the cpu
    gauge.saveCPUField(g, QUDA_CPU_FIELD_LOCATION);
#endif


    //Create a field UV which holds U*V.  Has the same similar
    //structure to V but double the number of spins so we can store
    //the four distinct block chiral multiplications in a single UV
    //computation.
    ColorSpinorParam UVparam(T.Vectors());
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    UVparam.nSpin *= 2; // so nSpin == 4
    cpuColorSpinorField uv(UVparam);

    calculateYcoarse(Y, X, Xinv, Yhat, uv, T, gauge, clover, cloverInv, kappa, mu, dirac, matpc);

  }
  
} //namespace quda
