#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <su3_project.cuh>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <index_helper.cuh>

#define  DOUBLE_TOL	1e-15
#define  SINGLE_TOL	2e-6

namespace quda {

#ifdef GPU_GAUGE_TOOLS

  template <typename Float, typename GaugeOr, typename GaugeDs>
  struct GaugeSTOUTArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4]; 
#endif
    GaugeOr origin;
    const Float rho;
    const Float tolerance;
    
    GaugeDs dest;

    GaugeSTOUTArg(GaugeOr &origin, GaugeDs &dest, const GaugeField &data, const Float rho, const Float tolerance) 
      : origin(origin), dest(dest), rho(rho), tolerance(tolerance) {
#ifdef MULTI_GPU
      for ( int dir = 0; dir < 4; ++dir ) {
        border[dir] = data.R()[dir];
        X[dir] = data.X()[dir] - border[dir] * 2;
      } 
#else
        for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir];
#endif
	threads = X[0]*X[1]*X[2]*X[3];
    }
  };


  template <typename Float, typename GaugeOr, typename GaugeDs, typename Float2>
  __host__ __device__ void computeStaple(GaugeSTOUTArg<Float,GaugeOr,GaugeDs>& arg, int idx, int parity, int dir, Matrix<Float2,3> &staple) {

    typedef Matrix<complex<Float>,3> Link;
      // compute spacetime dimensions and parity

    int X[4]; 
    for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

    int x[4];
    getCoords(x, idx, X, parity);
#ifdef MULTI_GPU
    for(int dr=0; dr<4; ++dr) {
         x[dr] += arg.border[dr];
         X[dr] += 2*arg.border[dr];
    }
#endif

    setZero(&staple);

    for (int mu=0; mu<3; mu++) {  // I believe most users won't want to include time staples in smearing
      if (mu == dir) {
        continue;
      }

      int nu = dir;

      {
        int dx[4] = {0, 0, 0, 0};
        Link U1, U2, U3, U4, tmpS;
        arg.origin.load((Float*)(U1.data),linkIndexShift(x,dx,X), mu, parity); 

        dx[mu]++;
        arg.origin.load((Float*)(U2.data),linkIndexShift(x,dx,X), nu, 1-parity); 

        dx[mu]--;
        dx[nu]++;
        arg.origin.load((Float*)(U3.data),linkIndexShift(x,dx,X), mu, 1-parity); 
   
        tmpS	= U1 * U2;
	tmpS	= tmpS * conj(U3);

	staple = staple + tmpS;

        dx[mu]--;
        dx[nu]--;
        arg.origin.load((Float*)(U1.data),linkIndexShift(x,dx,X), mu, 1-parity); 
        arg.origin.load((Float*)(U2.data),linkIndexShift(x,dx,X), nu, 1-parity); 

        dx[nu]++;
        arg.origin.load((Float*)(U3.data),linkIndexShift(x,dx,X), mu, parity); 

        tmpS	= conj(U1);
	tmpS	= tmpS * U2;
	tmpS	= tmpS * U3;

	staple = staple + tmpS;
      }
    }
  }

  template<typename Float, typename GaugeOr, typename GaugeDs>
    __global__ void computeSTOUTStep(GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg){
      int idx = threadIdx.x + blockIdx.x*blockDim.x;
      if(idx >= arg.threads) return;
      typedef complex<Float> Complex;
      typedef Matrix<complex<Float>,3> Link;

      int parity = 0;
      if(idx >= arg.threads/2) {
        parity = 1;
        idx -= arg.threads/2;
      }

      int X[4]; 
      for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

      int x[4];
      getCoords(x, idx, X, parity);
#ifdef MULTI_GPU
      for(int dr=0; dr<4; ++dr) {
           x[dr] += arg.border[dr];
           X[dr] += 2*arg.border[dr];
      }
#endif

      int dx[4] = {0, 0, 0, 0};
      for (int dir=0; dir < 3; dir++) {				//Only spatial dimensions are smeared
        Link U, UDag, Stap, Omega, OmegaDag, OmegaDiff, ODT, Q, exp_iQ, tmp1;
	Complex OmegaDiffTr;
	Complex i_2(0,0.5);

	//This function gets stap = S_{mu,nu} i.e., the staple of length 3,
        computeStaple<Float,GaugeOr,GaugeDs,Complex>(arg,idx,parity,dir,Stap);
	//
	// |- > -|
	// ^     v
	// |     |
	//          +  |     |
	//             v     ^
	//             |- > -|

	// Get link U
        arg.origin.load((Float*)(U.data),linkIndexShift(x,dx,X),dir,parity);

	//Compute Omega_{mu}=[Sum_{mu neq nu}rho_{mu,nu}C_{mu,nu}]*U_{mu}^dag

	//Get U^{\dagger}
	computeMatrixInverse(U,&UDag);
	
	//Compute \Omega = \rho * S * U^{\dagger}
	tmp1 = arg.rho * Stap;
	Omega = tmp1 * UDag;

	//Compute \Q_{mu} = i/2[Omega_{mu}^dag - Omega_{mu} 
	//                      - 1/3 Tr(Omega_{mu}^dag - Omega_{mu})]

	OmegaDag = conj(Omega);
	OmegaDiff = OmegaDag - Omega;

	Q = OmegaDiff;
	OmegaDiffTr = getTrace(OmegaDiff);
	OmegaDiffTr =  1.0/3.0 * OmegaDiffTr;

	//Matrix proportional to OmegaDiffTr
	setIdentity(&ODT);
	tmp1 = OmegaDiffTr * ODT;

	Q = Q - tmp1;
	Q = i_2 * Q;
	//Q is now defined.

	exponentiate_iQ(Q,&exp_iQ);
	U = exp_iQ * U;

        arg.dest.save((Float*)(U.data),linkIndexShift(x,dx,X), dir, parity); 
    }
  }

  template<typename Float, typename GaugeOr, typename GaugeDs>
    class GaugeSTOUT : Tunable {
      GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg;
      const QudaFieldLocation location;

      private:
      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool tuneSharedBytes() const { return false; } // Don't tune shared memory
      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
      unsigned int minThreads() const { return arg.threads; }

      public:
      GaugeSTOUT(GaugeSTOUTArg<Float,GaugeOr, GaugeDs> &arg, QudaFieldLocation location)
        : arg(arg), location(location) {}
      virtual ~GaugeSTOUT () {}

      void apply(const hipStream_t &stream){
        if (location == QUDA_CUDA_FIELD_LOCATION) {
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
          computeSTOUTStep<<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
        } else {
          errorQuda("CPU not supported yet\n");
          //computeSTOUTStepCPU(arg);
        }
      }

      TuneKey tuneKey() const {
        std::stringstream vol, aux;
        vol << arg.X[0] << "x";
        vol << arg.X[1] << "x";
        vol << arg.X[2] << "x";
        vol << arg.X[3];
        aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
        return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
      }

      long long flops() const { return (1)*6*arg.threads; }
      long long bytes() const { return (1)*6*arg.threads*sizeof(Float); } // Only correct if there is no link reconstruction

    }; // GaugeSTOUT

  template<typename Float,typename GaugeOr, typename GaugeDs>
    void STOUTStep(GaugeOr origin, GaugeDs dest, const GaugeField& dataOr, Float rho, QudaFieldLocation location) {
      if (dataOr.Precision() == QUDA_DOUBLE_PRECISION) {
        GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg(origin, dest, dataOr, rho, DOUBLE_TOL);
        GaugeSTOUT<Float,GaugeOr,GaugeDs> gaugeSTOUT(arg, location);
        gaugeSTOUT.apply(0);
      } else {
        GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg(origin, dest, dataOr, rho, SINGLE_TOL);
        GaugeSTOUT<Float,GaugeOr,GaugeDs> gaugeSTOUT(arg, location);
        gaugeSTOUT.apply(0);
      }
      hipDeviceSynchronize();
    }

  template<typename Float>
    void STOUTStep(GaugeField &dataDs, const GaugeField& dataOr, Float rho, QudaFieldLocation location) {

    if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GDs;

      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
      }
    } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_12){
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GDs;
      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
      }
    } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_8){
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GDs;
      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
    } else {
      errorQuda("Reconstruction type %d of destination gauge field not supported", dataDs.Reconstruct());
    }

  }

#endif

  void STOUTStep(GaugeField &dataDs, const GaugeField& dataOr, double rho, QudaFieldLocation location) {

#ifdef GPU_GAUGE_TOOLS

    if(dataOr.Precision() != dataDs.Precision()) {
      errorQuda("Oriign and destination fields must have the same precision\n");
    }

    if(dataDs.Precision() == QUDA_HALF_PRECISION){
      errorQuda("Half precision not supported\n");
    }

    if (!dataOr.isNative())
      errorQuda("Order %d with %d reconstruct not supported", dataOr.Order(), dataOr.Reconstruct());

    if (!dataDs.isNative())
      errorQuda("Order %d with %d reconstruct not supported", dataDs.Order(), dataDs.Reconstruct());

    if (dataDs.Precision() == QUDA_SINGLE_PRECISION){
      STOUTStep<float>(dataDs, dataOr, (float) rho, location);
    } else if(dataDs.Precision() == QUDA_DOUBLE_PRECISION) {
      STOUTStep<double>(dataDs, dataOr, rho, location);
    } else {
      errorQuda("Precision %d not supported", dataDs.Precision());
    }
    return;
#else
  errorQuda("Gauge tools are not build");
#endif
  }

}
