#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <typeinfo>
#include <vector>
#include <assert.h>

namespace quda {

  using namespace quda::colorspinor;

//ok for staggered: nSpin = 1 will work as well. Accessors do allow this case as well.

  // copy the null-space vectors into the V-field
  template <int nSpin, int nColor, int nVec, class V, class B>
  void fill(V &out, const B &in, int v) {
    for (int parity=0; parity<out.Nparity(); parity++) {
      for (int x_cb=0; x_cb<out.VolumeCB(); x_cb++) {
	for (int s=0; s<nSpin; s++) {
	  for (int c=0; c<nColor; c++) {
	    out(parity, x_cb, s, c, v) = in(parity, x_cb, s, c);
	  }
	}
      }
    }
  }

  template <typename Float, int nSpin, int nColor, int nVec, QudaFieldOrder order>
  void fillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B) {
    FieldOrderCB<Float,nSpin,nColor,nVec,order> vOrder(const_cast<ColorSpinorField&>(V));
    for (int v=0; v<nVec; v++) {
      FieldOrderCB<Float,nSpin,nColor,1,order> bOrder(const_cast<ColorSpinorField&>(*B[v]));
      fill<nSpin,nColor,nVec>(vOrder, bOrder, v);
    }
  }

  template <typename real, int nSpin, int nColor, int nVec, QudaFieldOrder order>
  class FillVLaunch : public Tunable {

    ColorSpinorField &V;
    const std::vector<ColorSpinorField*> &B;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

  public:
    FillVLaunch(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B) : V(V), B(B) {
      (V.Location() == QUDA_CPU_FIELD_LOCATION) ? strcpy(aux, "CPU") : strcpy(aux,"GPU");
    }
    virtual ~FillVLaunch() { }

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (V.Location() == QUDA_CPU_FIELD_LOCATION) {
	fillV<real,nSpin,nColor,nVec,order>(V,B);
      } else {
	errorQuda("Not implemented for GPU");
      }
    }

    bool advanceTuneParam(TuneParam &param) const { return false; }

    TuneKey tuneKey() const { return TuneKey(V.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return 0; }
    long long bytes() const { return 2*V.Bytes(); }
  };


  template <typename real, int nSpin, int nColor, int nVec, QudaFieldOrder order>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B) {
    FillVLaunch<real,nSpin,nColor,nVec,order> f(V,B);
    f.apply(0);
  }

//for staggered: this does not include factor 2 due to parity decomposition!

  template <typename Float, int nSpin, int nColor, QudaFieldOrder order>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (Nvec == 2) {
      FillV<Float,nSpin,nColor,2,order>(V,B);
    } else if (Nvec == 4) {
      FillV<Float,nSpin,nColor,4,order>(V,B);
    } else if (Nvec == 8) {
      FillV<Float,nSpin,nColor,8,order>(V,B);
    } else if (Nvec == 12) {
      FillV<Float,nSpin,nColor,12,order>(V,B);
    } else if (Nvec == 16) {
      FillV<Float,nSpin,nColor,16,order>(V,B);
    } else if (Nvec == 20) {
      FillV<Float,nSpin,nColor,20,order>(V,B);
    } else if (Nvec == 24) {
      FillV<Float,nSpin,nColor,24,order>(V,B);
    } else if (Nvec == 32) {
      FillV<Float,nSpin,nColor,32,order>(V,B);
    } else if (Nvec == 48) {
      FillV<Float,nSpin,nColor,48,order>(V,B);
    } else {
      errorQuda("Unsupported Nvec %d", Nvec);
    }
  }

//ok for 2-cycle multigrid, must be extended for more complicated version.

  template <typename Float, int nSpin, QudaFieldOrder order>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (B[0]->Ncolor()*Nvec != V.Ncolor()) errorQuda("Something wrong here");

    if (B[0]->Ncolor() == 2) {
      FillV<Float,nSpin,2,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 3) {
      FillV<Float,nSpin,3,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 8) {
      FillV<Float,nSpin,8,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 16) {
      FillV<Float,nSpin,16,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 24) {
      FillV<Float,nSpin,24,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 32) {
      FillV<Float,nSpin,32,order>(V,B,Nvec);
    } else {
      errorQuda("Unsupported nColor %d", B[0]->Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder order>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (V.Nspin() == 4) {
      FillV<Float,4,order>(V,B,Nvec);
    } else if (V.Nspin() == 2) {
      FillV<Float,2,order>(V,B,Nvec);
#ifdef GPU_STAGGERED_DIRAC
    } else if (V.Nspin() == 1) {
      FillV<Float,1,order>(V,B,Nvec);
#endif
    } else {
      errorQuda("Unsupported nSpin %d", V.Nspin());
    }
  }

  template <typename Float>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (V.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      FillV<Float,QUDA_FLOAT2_FIELD_ORDER>(V,B,Nvec);
    } else if (V.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      FillV<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(V,B,Nvec);
    } else {
      errorQuda("Unsupported field type %d", V.FieldOrder());
    }
  }

  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (V.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      FillV<double>(V,B,Nvec);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (V.Precision() == QUDA_SINGLE_PRECISION) {
      FillV<float>(V,B,Nvec);
    } else {
      errorQuda("Unsupported precision %d", V.Precision());
    }
  }

  // Creates a block-ordered version of a ColorSpinorField
  // N.B.: Only works for the V field, as we need to block spin.
  template <bool toBlock, int nVec, class Complex, class FieldOrder>
  void blockOrderV(Complex *out, FieldOrder &in,
		   const int *geo_map, const int *geo_bs, int spin_bs,
		   const cpuColorSpinorField &V) {
    //printfQuda("in.Ncolor = %d\n", in.Ncolor());
    int nSpin_coarse = in.Nspin() / spin_bs; // this is number of chiral blocks

    //Compute the size of each block
    int geoBlockSize = 1;
    for (int d=0; d<in.Ndim(); d++) geoBlockSize *= geo_bs[d];
    int blockSize = geoBlockSize * in.Ncolor() * spin_bs; // blockSize includes internal dof

    int x[QUDA_MAX_DIM]; // global coordinates
    int y[QUDA_MAX_DIM]; // local coordinates within a block (full site ordering)

    int checkLength = in.Nparity() * in.VolumeCB() * in.Ncolor() * in.Nspin() * in.Nvec();
    int *check = new int[checkLength];
    int count = 0;

    // Run through the fine grid and do the block ordering
    for (int parity = 0; parity<in.Nparity(); parity++) {
      for (int x_cb=0; x_cb<in.VolumeCB(); x_cb++) {
	int i = parity*in.VolumeCB() + x_cb;

	// Get fine grid coordinates
	V.LatticeIndex(x, i);
	
	//Compute the geometric offset within a block 
	// (x fastest direction, t is slowest direction, non-parity ordered)
	int blockOffset = 0;
	for (int d=in.Ndim()-1; d>=0; d--) {
	  y[d] = x[d]%geo_bs[d];
	  blockOffset *= geo_bs[d];
	  blockOffset += y[d];
	}
	
	//Take the block-ordered offset from the coarse grid offset (geo_map) 
	int offset = geo_map[i]*nSpin_coarse*nVec*geoBlockSize*in.Ncolor()*spin_bs;
	
	for (int v=0; v<in.Nvec(); v++) {
	  for (int s=0; s<in.Nspin(); s++) {
	    for (int c=0; c<in.Ncolor(); c++) {
	      
	      int chirality = s / spin_bs; // chirality is the coarse spin
	      int blockSpin = s % spin_bs; // the remaining spin dof left in each block
	      
	      int index = offset +                                              // geo block
		chirality * nVec * geoBlockSize * spin_bs * in.Ncolor() + // chiral block
	                       v * geoBlockSize * spin_bs * in.Ncolor() + // vector
	                            blockOffset * spin_bs * in.Ncolor() + // local geometry
	                                          blockSpin*in.Ncolor() + // block spin
	                                                                   c;   // color

	      if (toBlock) out[index] = in(parity, x_cb, s, c, v); // going to block order
	      else in(parity, x_cb, s, c, v) = out[index]; // coming from block order
	    
	      check[count++] = index;
	    }
	  }
	}
      }

      //printf("blockOrderV done %d / %d\n", i, in.Volume());
    }
    
    if (count != checkLength) {
      errorQuda("Number of elements packed %d does not match expected value %d nvec=%d nspin=%d ncolor=%d", 
		count, checkLength, in.Nvec(), in.Nspin(), in.Ncolor());
    }

    /*
    // need non-quadratic check
    for (int i=0; i<checkLength; i++) {
      for (int j=0; j<i; j++) {
      if (check[i] == check[j]) errorQuda("Collision detected in block ordering\n");
      }
    }
    */
    delete []check;
  }


  // Creates a block-ordered version of a ColorSpinorField, with parity blocking (for staggered fields)
  // N.B.: same as above but parity are separated.
  template <bool toBlock, int nVec, class Complex, class FieldOrder>
  void blockCBOrderV(Complex *out, FieldOrder &in,
		     const int *geo_map, const int *geo_bs, int spin_bs,
		     const cpuColorSpinorField &V) {
    //Compute the size of each block
    int geoBlockSize = 1;
    for (int d=0; d<in.Ndim(); d++) geoBlockSize *= geo_bs[d];
    int blockSize = geoBlockSize * in.Ncolor(); // blockSize includes internal dof

    int x[QUDA_MAX_DIM]; // global coordinates
    int y[QUDA_MAX_DIM]; // local coordinates within a block (full site ordering)

    int checkLength = in.Nparity() * in.VolumeCB() * in.Ncolor() * in.Nvec();
    int *check = new int[checkLength];
    int count = 0;

    // Run through the fine grid and do the block ordering
    for (int parity = 0; parity<in.Nparity(); parity++) {
      for (int x_cb=0; x_cb<in.VolumeCB(); x_cb++) {
	int i = parity*in.VolumeCB() + x_cb;

	// Get fine grid coordinates
	V.LatticeIndex(x, i);

	//Compute the geometric offset within a block 
	// (x fastest direction, t is slowest direction, non-parity ordered)
	int blockOffset = 0;
	for (int d=in.Ndim()-1; d>=0; d--) {
	  y[d] = x[d]%geo_bs[d];
	  blockOffset *= geo_bs[d];
	  blockOffset += y[d];
	}

	//Take the block-ordered offset from the coarse grid offset (geo_map) 
	//A.S.: geo_map introduced for the full site ordering, so ok to use it for the offset
	int offset = geo_map[i]*nVec*geoBlockSize*in.Ncolor();

	const int s = 0;

	for (int v=0; v<in.Nvec(); v++) {
	  for (int c=0; c<in.Ncolor(); c++) {

	    int chirality = (x[0]+x[1]+x[2]+x[3])%2; // chirality is the fine-grid parity flag

	    int index = offset +                                // geo block
	      chirality * nVec * geoBlockSize * in.Ncolor() + // chiral block
	                     v * geoBlockSize * in.Ncolor() + // vector
	                          blockOffset * in.Ncolor() + // local geometry
	                                                       c;   // color

	    if (toBlock) out[index] = in(parity, x_cb, s, c, v); // going to block order
	    else in(parity, x_cb, s, c, v) = out[index]; // coming from block order

	    check[count++] = index;
	  }
	}

	//printf("blockOrderV done %d / %d\n", i, in.Volume());
      } // x_cb
    } // parity

    if (count != checkLength) {
      errorQuda("Number of elements packed %d does not match expected value %d nvec=%d ncolor=%d", 
		count, checkLength, in.Nvec(), in.Ncolor());
    }

    delete []check;
  }




  // Orthogonalise the nc vectors v[] of length n
  // this assumes the ordering v[(b * Nvec + v) * blocksize + i]

  template <typename sumFloat, typename Float, int N>
  void blockGramSchmidt(complex<Float> *v, int nBlocks, int blockSize) {
    
    for (int b=0; b<nBlocks; b++) {
      for (int jc=0; jc<N; jc++) {
      
	for (int ic=0; ic<jc; ic++) {
	  // Calculate dot product.
	  complex<Float> dot = 0.0;
	  for (int i=0; i<blockSize; i++) 
	    dot += conj(v[(b*N+ic)*blockSize+i]) * v[(b*N+jc)*blockSize+i];
	  
	  // Subtract the blocks to orthogonalise
	  for (int i=0; i<blockSize; i++) 
	    v[(b*N+jc)*blockSize+i] -= dot * v[(b*N+ic)*blockSize+i];
	}
	
	// Normalize the block
	// nrm2 is pure real, but need to use Complex because of template.
        sumFloat nrm2 = 0.0;
	for (int i=0; i<blockSize; i++) nrm2 += norm(v[(b*N+jc)*blockSize+i]);
	sumFloat scale = nrm2 > 0.0 ? 1.0/sqrt(nrm2) : 0.0;
	for (int i=0; i<blockSize; i++) v[(b*N+jc)*blockSize+i] *= scale;
      }

      /*      
      for (int jc=0; jc<N; jc++) {
        complex<sumFloat> nrm2 = 0.0;
        for(int i=0; i<blockSize; i++) nrm2 += norm(v[(b*N+jc)*blockSize+i]);
	//printfQuda("block = %d jc = %d nrm2 = %f\n", b, jc, nrm2.real());
      }
      */

      //printf("blockGramSchmidt done %d / %d\n", b, nBlocks);
    }

  }

  template <typename sumType, typename real, int N>
  class BlockGramSchmidt : public Tunable {

    complex<real> *v;
    int nBlock;
    int blockSize;
    const ColorSpinorField &meta;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

  public:
    BlockGramSchmidt(complex<real> *v, int nBlock, int blockSize, const ColorSpinorField &meta)
      : v(v), nBlock(nBlock), blockSize(blockSize), meta(meta) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) sprintf(aux, "nBlock=%d,blockSize=%d,CPU", nBlock, blockSize);
      else sprintf(aux, "nBlock=%d,blockSize=%d,GPU", nBlock, blockSize);
    }

    virtual ~BlockGramSchmidt() { }

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	blockGramSchmidt<sumType, real, N>(v, nBlock, blockSize);
      } else {
	errorQuda("Not implemented for GPU");
      }
    }

    bool advanceTuneParam(TuneParam &param) const { return false; }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return nBlock * N * ((N-1) * (8l + 8l) + 2l) * blockSize; }
    long long bytes() const { return 2*meta.Bytes(); }
  };

  template <bool toBlock, int N, typename real, typename Order>
  class BlockOrderV : public Tunable {

    complex<real> *vBlock;
    Order &vOrder;
    const int *geo_map;
    const int *geo_bs;
    int spin_bs;
    const ColorSpinorField &V;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

  public:
    BlockOrderV(complex<real> *vBlock, Order &vOrder, const int *geo_map, const int *geo_bs, int spin_bs, const ColorSpinorField &V)
      : vBlock(vBlock), vOrder(vOrder), geo_map(geo_map), geo_bs(geo_bs), spin_bs(spin_bs), V(V) {
      (V.Location() == QUDA_CPU_FIELD_LOCATION) ? strcpy(aux, "CPU") : strcpy(aux,"GPU");
    }

    virtual ~BlockOrderV() { }

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (V.Location() == QUDA_CPU_FIELD_LOCATION) {
	blockOrderV<toBlock,N,complex<real>,Order>(vBlock,vOrder,geo_map,geo_bs,spin_bs,V);
      } else {
	errorQuda("Not implemented for GPU");
      }
    }

    bool advanceTuneParam(TuneParam &param) const { return false; }

    TuneKey tuneKey() const { return TuneKey(V.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return 0; }
    long long bytes() const { return 2*V.Bytes(); }
  };


  template<typename Float, int nSpin, int nColor, int nVec, QudaFieldOrder order>
  void BlockOrthogonalize(ColorSpinorField &V, const int *geo_bs, const int *geo_map, int spin_bs) {
    complex<Float> *Vblock = new complex<Float>[V.Volume()*V.Nspin()*V.Ncolor()];

    typedef FieldOrderCB<Float,nSpin,nColor,nVec,order> VectorField;
    VectorField vOrder(const_cast<ColorSpinorField&>(V));

    int geo_blocksize = 1;
    for (int d = 0; d < V.Ndim(); d++) geo_blocksize *= geo_bs[d];

    int blocksize = geo_blocksize * vOrder.Ncolor() * spin_bs; 
    int chiralBlocks = (V.Nspin() == 1) ? 2 : vOrder.Nspin() / spin_bs; //always 2 for staggered. 
    int numblocks = (V.Volume()/geo_blocksize) * chiralBlocks;
    if (V.Nspin() == 1) blocksize /= chiralBlocks; //for staggered chiral block size is a parity block size
    
    printfQuda("Block Orthogonalizing %d blocks of %d length and width %d\n", numblocks, blocksize, nVec);

    BlockOrderV<true,nVec,Float,VectorField> reorder(Vblock, vOrder, geo_map, geo_bs, spin_bs, V);
    reorder.apply(0);

    BlockGramSchmidt<double,Float,nVec> ortho(Vblock, numblocks, blocksize, V);
    ortho.apply(0);

    BlockOrderV<false,nVec,Float,VectorField> reset(Vblock, vOrder, geo_map, geo_bs, spin_bs, V);
    reset.apply(0);

    delete []Vblock;
  }


  template<typename Float, int nSpin, int nColor, QudaFieldOrder order>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, const int *geo_bs, const int *geo_map, int spin_bs) {
    if (Nvec == 2) {
      BlockOrthogonalize<Float,nSpin,nColor,2,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 4) {
      BlockOrthogonalize<Float,nSpin,nColor,4,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 8) {
      BlockOrthogonalize<Float,nSpin,nColor,8,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 12) {
      BlockOrthogonalize<Float,nSpin,nColor,12,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 16) {
      BlockOrthogonalize<Float,nSpin,nColor,16,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 20) {
      BlockOrthogonalize<Float,nSpin,nColor,20,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 24) {
      BlockOrthogonalize<Float,nSpin,nColor,24,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 32) {
      BlockOrthogonalize<Float,nSpin,nColor,32,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 48) {
      BlockOrthogonalize<Float,nSpin,nColor,48,order>(V, geo_bs, geo_map, spin_bs);
    } else {
      errorQuda("Unsupported nVec %d\n", Nvec);
    }
  }

  template<typename Float, int nSpin, QudaFieldOrder order>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Ncolor()/Nvec == 3) {
      BlockOrthogonalize<Float,nSpin,3,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 2) {
      BlockOrthogonalize<Float,nSpin,2,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 8) {
      BlockOrthogonalize<Float,nSpin,8,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 16) {
      BlockOrthogonalize<Float,nSpin,16,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 24) {
      BlockOrthogonalize<Float,nSpin,24,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 32) {
      BlockOrthogonalize<Float,nSpin,32,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 48) {
      BlockOrthogonalize<Float,nSpin,48,order>(V, Nvec, geo_bs, geo_map, spin_bs); //for staggered, even-odd blocking presumed
    }  
    else {
      errorQuda("Unsupported nColor %d\n", V.Ncolor()/Nvec);
    }
  }

  template<typename Float, QudaFieldOrder order>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Nspin() == 4) {
      BlockOrthogonalize<Float,4,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if(V.Nspin() ==2) {
      BlockOrthogonalize<Float,2,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    } 
    else if (V.Nspin() == 1) {
      BlockOrthogonalize<Float,1,order>(V, Nvec, geo_bs, geo_map, 1);
    }
    else {
      errorQuda("Unsupported nSpin %d\n", V.Nspin());
    }
  }

  template<typename Float>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
  if (V.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      BlockOrthogonalize<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else {
      errorQuda("Unsupported field order %d\n", V.FieldOrder());
    }
  }

  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      BlockOrthogonalize<double>(V, Nvec, geo_bs, geo_map, spin_bs);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (V.Precision() == QUDA_SINGLE_PRECISION) {
      BlockOrthogonalize<float>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else {
      errorQuda("Unsupported precision %d\n", V.Precision());
    }
  }

} // namespace quda
