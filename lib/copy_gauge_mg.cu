#include <gauge_field_order.h>

#define FINE_GRAINED_ACCESS

#include <copy_gauge_helper.cuh>

namespace quda {
  
  template <typename FloatOut, typename FloatIn, int length, typename InOrder>
  void copyGaugeMG(const InOrder &inOrder, GaugeField &out, QudaFieldLocation location,
		   FloatOut *Out, FloatOut **outGhost, int type) {
    if (out.Reconstruct() != QUDA_RECONSTRUCT_NO) 
      errorQuda("Reconstruct type %d not supported", out.Reconstruct());

    int faceVolumeCB[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) faceVolumeCB[i] = out.SurfaceCB(i) * out.Nface(); 
    if (out.isNative()) {

#ifdef FINE_GRAINED_ACCESS
      typedef typename gauge::FieldOrder<FloatOut,Ncolor(length),1,QUDA_FLOAT2_GAUGE_ORDER> G;
      copyGauge<FloatOut,FloatIn,length>(G(out,(void*)Out,(void**)outGhost), inOrder, out.Volume(), faceVolumeCB,
					 out.Ndim(), out.Geometry(), out, location, type);
#else
      typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_NO,length>::type G;
      copyGauge<FloatOut,FloatIn,length>
	(G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	 out.Ndim(), out.Geometry(), out, location, type);
#endif

    } else if (out.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE

#ifdef FINE_GRAINED_ACCESS
      typedef typename gauge::FieldOrder<FloatOut,Ncolor(length),1,QUDA_QDP_GAUGE_ORDER> G;
      copyGauge<FloatOut,FloatIn,length>(G(out,(void*)Out,(void**)outGhost), inOrder, out.Volume(),
					 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      typedef typename QDPOrder<FloatOut,length> G;
      copyGauge<FloatOut,FloatIn,length>(G(out, Out, outGhost), inOrder, out.Volume(),
					 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#endif


#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", out.Order());
    }

  }

  template <typename FloatOut, typename FloatIn, int length>
    void copyGaugeMG(GaugeField &out, const GaugeField &in, QudaFieldLocation location,
		     FloatOut *Out, FloatIn *In, FloatOut **outGhost, FloatIn **inGhost, int type) {

    if (in.Reconstruct() != QUDA_RECONSTRUCT_NO) 
      errorQuda("Reconstruct type %d not supported", in.Reconstruct());

    // reconstruction only supported on FloatN fields currently
    if (in.isNative()) {      
#ifdef FINE_GRAINED_ACCESS
      typedef typename gauge::FieldOrder<FloatIn,Ncolor(length),1,QUDA_FLOAT2_GAUGE_ORDER> G;
      copyGaugeMG<FloatOut,FloatIn,length> (G(const_cast<GaugeField&>(in),(void*)In,(void**)inGhost), out, location, Out, outGhost, type);
#else
      typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_NO,length>::type G;
      copyGaugeMG<FloatOut,FloatIn,length> (G(in, In,inGhost), out, location, Out, outGhost, type);
#endif
    } else if (in.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE

#ifdef FINE_GRAINED_ACCESS
      typedef typename gauge::FieldOrder<FloatIn,Ncolor(length),1,QUDA_QDP_GAUGE_ORDER> G;
      copyGaugeMG<FloatOut,FloatIn,length>(G(const_cast<GaugeField&>(in),(void*)In,(void**)inGhost), out, location, Out, outGhost, type);
#else
      typedef typename QDPOrder<FloatIn,length> G;
      copyGaugeMG<FloatOut,FloatIn,length>(G(in, In, inGhost), out, location, Out, outGhost, type);
#endif

#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", in.Order());
    }

  }

  template <typename FloatOut, typename FloatIn>
  void copyGaugeMG(GaugeField &out, const GaugeField &in, QudaFieldLocation location, FloatOut *Out, 
		   FloatIn *In, FloatOut **outGhost, FloatIn **inGhost, int type) {

#ifdef GPU_MULTIGRID
    if (in.Ncolor() == 4) {
      const int Nc = 4;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else  if (in.Ncolor() == 8) {
      const int Nc = 8;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else  if (in.Ncolor() == 16) {
      const int Nc = 16;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else  if (in.Ncolor() == 24) {
      const int Nc = 24;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else  if (in.Ncolor() == 32) {
      const int Nc = 32;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else  if (in.Ncolor() == 40) {
      const int Nc = 40;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else  if (in.Ncolor() == 48) {
      const int Nc = 48;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else  if (in.Ncolor() == 56) {
      const int Nc = 56;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else  if (in.Ncolor() == 64) {
      const int Nc = 64;
      copyGaugeMG<FloatOut,FloatIn,2*Nc*Nc>(out, in, location, Out, In, outGhost, inGhost, type);
    } else 
#endif // GPU_MULTIGRID
    {
      errorQuda("Unsupported number of colors; out.Nc=%d, in.Nc=%d", out.Ncolor(), in.Ncolor());
    }
  }

  // this is the function that is actually called, from here on down we instantiate all required templates
  void copyGenericGaugeMG(GaugeField &out, const GaugeField &in, QudaFieldLocation location,
			  void *Out, void *In, void **ghostOut, void **ghostIn, int type) {
    if (out.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      if (in.Precision() == QUDA_DOUBLE_PRECISION) {
	copyGaugeMG(out, in, location, (double*)Out, (double*)In, (double**)ghostOut, (double**)ghostIn, type);
      } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
	copyGaugeMG(out, in, location, (double*)Out, (float*)In, (double**)ghostOut, (float**)ghostIn, type);
      } else {
	errorQuda("Precision %d not supported", in.Precision());
      }
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (out.Precision() == QUDA_SINGLE_PRECISION) {
      if (in.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
	copyGaugeMG(out, in, location, (float*)Out, (double*)In, (float**)ghostOut, (double**)ghostIn, type);
#else
	errorQuda("Double precision multigrid has not been enabled");
#endif
      } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
	copyGaugeMG(out, in, location, (float*)Out, (float*)In, (float**)ghostOut, (float**)ghostIn, type);
      } else {
	errorQuda("Precision %d not supported", in.Precision());
      }
    } else {
      errorQuda("Precision %d not supported", out.Precision());
    } 
  } 



} // namespace quda
