#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cstring> // needed for memset



#include <tune_quda.h>
#include <typeinfo>

#include <quda_internal.h>
#include <float_vector.h>
#include <blas_quda.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <face_quda.h> // this is where the MPI / QMP depdendent code is

#define checkSpinor(a, b)						\
  {									\
    if (a.Precision() != b.Precision())					\
      errorQuda("precisions do not match: %d %d", a.Precision(), b.Precision()); \
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length()); \
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

#define checkLength(a, b)						\
  {									\
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length()); \
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

namespace quda {

  namespace blas {

#define BLAS_SPINOR // do not include ghost functions in Spinor class to reduce parameter space overhead
#include <texture.h>

    unsigned long long flops;
    unsigned long long bytes;

    void zero(ColorSpinorField &a) {
      if (typeid(a) == typeid(cudaColorSpinorField)) {
	static_cast<cudaColorSpinorField&>(a).zero();
      } else {
	static_cast<cpuColorSpinorField&>(a).zero();
      }
    }

    static hipStream_t *blasStream;

    static struct {
      const char *vol_str;
      const char *aux_str;
      char aux_tmp[TuneKey::aux_n];
    } blasStrings;

    void initReduce();
    void endReduce();

    void init()
    {
      blasStream = &streams[Nstream-1];
      initReduce();
    }

    void end(void)
    {
      endReduce();
    }

    hipStream_t* getStream() { return blasStream; }

#include <blas_core.cuh>

#include <blas_core.h>
#include <blas_mixed_core.h>
#include <multi_blas_core.cuh>
#include <multi_blas_core.h>


    template <typename Float2, typename FloatN>
    struct BlasFunctor {

      //! pre-computation routine before the main loop
      virtual __device__ __host__ void init() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) = 0;
    };

    template <int NXZ, typename Float2, typename FloatN>
    struct MultiBlasFunctor {

      //! pre-computation routine before the main loop
      virtual __device__ __host__ void init() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j) = 0;
    };

    /**
       Functor to perform the operation y = a*x + b*y
    */
    template <typename Float2, typename FloatN>
    struct axpby_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      axpby_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y = a.x*x + b.x*y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 3; } //! flops per element
    };

    void axpby(const double &a, ColorSpinorField &x, const double &b, ColorSpinorField &y) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpby_,0,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
				       x, y, x, x);
      } else {
	blasCuda<axpby_,0,1,0,0>(make_double2(a, 0.0), make_double2(b, 0.0), make_double2(0.0, 0.0),
				 x, y, x, x);
      }
    }

    /**
       Functor to perform the operation y += x
    */
    template <typename Float2, typename FloatN>
    struct xpy_ : public BlasFunctor<Float2,FloatN> {
      xpy_(const Float2 &a, const Float2 &b, const Float2 &c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y += x ; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void xpy(ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<xpy_,0,1,0,0>(make_double2(1.0, 0.0), make_double2(1.0, 0.0),
			     make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to perform the operation y += a*x
    */
    template <typename Float2, typename FloatN>
    struct axpy_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      axpy_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y = a.x*x + y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    void axpy(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpy_,0,1,0,0>(make_double2(a,0.0), make_double2(1.0,0.0), make_double2(0.0,0.0),
				       x, y, x, x);
      } else {
	blasCuda<axpy_,0,1,0,0>(make_double2(a, 0.0), make_double2(1.0, 0.0), make_double2(0.0, 0.0),
			       x, y, x, x);
      }
    }

    /**
       Functor to perform the operation y = x + a*y
    */
    template <typename Float2, typename FloatN>
    struct xpay_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      xpay_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y = x + a.x*y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    void xpay(ColorSpinorField &x, const double &a, ColorSpinorField &y) {
      blasCuda<xpay_,0,1,0,0>(make_double2(a,0.0), make_double2(0.0, 0.0), make_double2(0.0, 0.0),
			     x, y, x, x);
    }

    /**
       Functor to perform the operation y -= x;
    */
    template <typename Float2, typename FloatN>
    struct mxpy_ : public BlasFunctor<Float2,FloatN> {
      mxpy_(const Float2 &a, const Float2 &b, const Float2 &c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y -= x; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void mxpy(ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<mxpy_,0,1,0,0>(make_double2(1.0, 0.0), make_double2(1.0, 0.0),
			     make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to perform the operation x *= a
    */
    template <typename Float2, typename FloatN>
    struct ax_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      ax_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { x *= a.x; }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void ax(const double &a, ColorSpinorField &x) {
      blasCuda<ax_,1,0,0,0>(make_double2(a, 0.0), make_double2(0.0, 0.0),
			   make_double2(0.0, 0.0), x, x, x, x);
    }

    /**
       Functor to perform the operation y += a * x  (complex-valued)
    */

    __device__ __host__ void _caxpy(const float2 &a, const float4 &x, float4 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
      y.z += a.x*x.z; y.z -= a.y*x.w;
      y.w += a.y*x.z; y.w += a.x*x.w;
    }

    __device__ __host__ void _caxpy(const float2 &a, const float2 &x, float2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }

    __device__ __host__ void _caxpy(const double2 &a, const double2 &x, double2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }

    template <typename Float2, typename FloatN>
    struct caxpy_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      caxpy_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    void caxpy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<caxpy_,0,1,0,0>(make_double2(real(a),imag(a)), make_double2(0.0, 0.0),
			       make_double2(0.0, 0.0), x, y, x, x);
    }

    template<int NXZ, typename Float2, typename FloatN>
    struct multicaxpy_ : public MultiBlasFunctor<NXZ, Float2, FloatN> {
      const int NYW;
      multicaxpy_(const Complex *a, int NYW) : NYW(NYW) { }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j)
      {
#ifdef __CUDA_ARCH__
	Float2 *a = reinterpret_cast<Float2*>(Amatrix_d); // fetch coefficient matrix from constant memory
	_caxpy(a[MAX_MULTI_BLAS_N*j+i], x, y);
#else
	Float2 *a = reinterpret_cast<Float2*>(Amatrix_h);
	_caxpy(a[NYW*j+i], x, y);
#endif
      }
      int streams() { return 2*NYW + NXZ*NYW; } //! total number of input and output streams
      int flops() { return 4*NXZ*NYW; } //! flops per real element
    };

    void caxpy(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      switch (x.size()) {
      case 1:
	multiblasCuda<1,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 2:
	multiblasCuda<2,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 3:
	multiblasCuda<3,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 4:
	multiblasCuda<4,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 5:
	multiblasCuda<5,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 6:
	multiblasCuda<6,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 7:
	multiblasCuda<7,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 8:
	multiblasCuda<8,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 9:
	multiblasCuda<9,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 10:
	multiblasCuda<10,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 11:
	multiblasCuda<11,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 12:
	multiblasCuda<12,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 13:
	multiblasCuda<13,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 14:
	multiblasCuda<14,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 15:
	multiblasCuda<15,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      case 16:
	multiblasCuda<16,multicaxpy_,0,1,0,0>(a, 0, 0, x, y, x, y);
        break;
      default:
	// split the problem in half and recurse
	const Complex *a0 = &a[0];
	const Complex *a1 = &a[x.size()*y.size()/2];

	std::vector<ColorSpinorField*> x0(x.begin(), x.begin() + x.size()/2);
	std::vector<ColorSpinorField*> x1(x.begin() + x.size()/2, x.end());

	caxpy(a0, x0, y);
	caxpy(a1, x1, y);
      }
    }

    void caxpy(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy(a, x.Components(), y.Components()); }

    /**
       Functor to perform the operation y = a*x + b*y  (complex-valued)
    */

    __device__ __host__ void _caxpby(const float2 &a, const float4 &x, const float2 &b, float4 &y)
    { float4 yy;
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;
      yy.z = a.x*x.z; yy.z -= a.y*x.w; yy.z += b.x*y.z; yy.z -= b.y*y.w;
      yy.w = a.y*x.z; yy.w += a.x*x.w; yy.w += b.y*y.z; yy.w += b.x*y.w;
      y = yy; }

    __device__ __host__ void _caxpby(const float2 &a, const float2 &x, const float2 &b, float2 &y)
    { float2 yy;
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;
      y = yy; }

    __device__ __host__ void _caxpby(const double2 &a, const double2 &x, const double2 &b, double2 &y)
    { double2 yy;
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;
      y = yy; }

    template <typename Float2, typename FloatN>
    struct caxpby_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpby_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpby(a, x, b, y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 7; } //! flops per element
    };

    void caxpby(const Complex &a, ColorSpinorField &x, const Complex &b, ColorSpinorField &y) {
      blasCuda<caxpby_,0,1,0,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)),
			       make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to performs the operation z[i] = x[i] + a*y[i] + b*z[i]
    */

    __device__ __host__ void _cxpaypbz(const float4 &x, const float2 &a, const float4 &y, const float2 &b, float4 &z) {
      float4 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      zz.z = x.z + a.x*y.z; zz.z -= a.y*y.w; zz.z += b.x*z.z; zz.z -= b.y*z.w;
      zz.w = x.w + a.y*y.z; zz.w += a.x*y.w; zz.w += b.y*z.z; zz.w += b.x*z.w;
      z = zz;
    }

    __device__ __host__ void _cxpaypbz(const float2 &x, const float2 &a, const float2 &y, const float2 &b, float2 &z) {
      float2 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      z = zz;
    }

    __device__ __host__ void _cxpaypbz(const double2 &x, const double2 &a, const double2 &y, const double2 &b, double2 &z) {
      double2 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      z = zz;
    }

    template <typename Float2, typename FloatN>
    struct cxpaypbz_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      cxpaypbz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _cxpaypbz(x, a, y, b, z); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void cxpaypbz(ColorSpinorField &x, const Complex &a, ColorSpinorField &y,
		  const Complex &b, ColorSpinorField &z) {
      blasCuda<cxpaypbz_,0,0,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)),
				 make_double2(0.0, 0.0), x, y, z, z);
    }

    /**
       Functor performing the operations: y[i] = a*x[i] + y[i]; x[i] = b*z[i] + c*x[i]
    */
    template <typename Float2, typename FloatN>
    struct axpyBzpcx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      const Float2 c;
      axpyBzpcx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b), c(c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y += a.x*x; x = b.x*z + c.x*x; }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 5; } //! flops per element
    };

    void axpyBzpcx(const double &a, ColorSpinorField& x, ColorSpinorField& y, const double &b,
		   ColorSpinorField& z, const double &c) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpyBzpcx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0),
					    make_double2(c,0.0),	x, y, z, x);
      } else {
	// swap arguments around
	blasCuda<axpyBzpcx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0),
				     make_double2(c,0.0), x, y, z, x);
      }
    }


    template<int NXZ, typename Float2, typename FloatN>
    struct multi_axpyBzpcx_ : public MultiBlasFunctor<NXZ, Float2, FloatN> {
      const int NYW;
      multi_axpyBzpcx_(const Complex *a, int NYW) : NYW(NYW) { }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j)
      {
#ifdef __CUDA_ARCH__
	// fetch coefficient arrays from constant memory
	Float2 *a = reinterpret_cast<Float2*>(Amatrix_d);
	Float2 *b = reinterpret_cast<Float2*>(Bmatrix_d);
	Float2 *c = reinterpret_cast<Float2*>(Cmatrix_d);
#else
	Float2 *a = reinterpret_cast<Float2*>(Amatrix_h);
	Float2 *b = reinterpret_cast<Float2*>(Bmatrix_h);
	Float2 *c = reinterpret_cast<Float2*>(Cmatrix_h);
#endif
	w += a[i].x * y;
	y = b[i].x * x + c[i].x * y;
      }
      int streams() { return 4*NYW + NXZ; } //! total number of input and output streams
      int flops() { return 5*NXZ*NYW; } //! flops per real element
    };

    void axpyBzpcx(const double *a_, std::vector<ColorSpinorField*> &x_, std::vector<ColorSpinorField*> &y_,
		   const double *b_, ColorSpinorField &z_, const double *c_) {

      // swizzle order since we are writing to x_ and y_, but the
      // multi-blas only allow writing to y and w, and moreover the
      // block width of y and w must match, and x and z must match.
      std::vector<ColorSpinorField*> &y = x_;
      std::vector<ColorSpinorField*> &w = y_;

      // wrap a container around the third solo vector
      std::vector<ColorSpinorField*> x;
      x.push_back(&z_);

      Complex *a = new Complex[y.size()];
      Complex *b = new Complex[y.size()];
      Complex *c = new Complex[y.size()];

      for (unsigned int i=0; i<y.size(); i++) { a[i] = Complex(a_[i]); b[i] = Complex(b_[i]); c[i] = Complex(c_[i]); }

      multiblasCuda<1,multi_axpyBzpcx_,0,1,0,1>(a, b, c, x, y, x, w);

      delete []a;
      delete []b;
      delete []c;
    }


    /**
       Functor performing the operations: y[i] = a*x[i] + y[i]; x[i] = z[i] + b*x[i]
    */
    template <typename Float2, typename FloatN>
    struct axpyZpbx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      axpyZpbx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y += a.x*x; x = z + b.x*x; }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    void axpyZpbx(const double &a, ColorSpinorField& x, ColorSpinorField& y,
		  ColorSpinorField& z, const double &b) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpyZpbx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
					   x, y, z, x);
      } else {
	// swap arguments around
	blasCuda<axpyZpbx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
				    x, y, z, x);
      }
    }

    /**
       Functor performing the operations z[i] = a*x[i] + b*y[i] + z[i] and y[i] -= b*w[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypzYmbw_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpbypzYmbw_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, z); _caxpy(b, y, z); _caxpy(-b, w, y); }

      static int streams() { return 6; } //! total number of input and output streams
      static int flops() { return 12; } //! flops per element
    };

    void caxpbypzYmbw(const Complex &a, ColorSpinorField &x, const Complex &b,
		      ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w) {
      blasCuda<caxpbypzYmbw_,0,1,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)),
				     make_double2(0.0,0.0), x, y, z, w);
    }

    /**
       Functor performing the operation y[i] += a*b*x[i], x[i] *= a
    */
    template <typename Float2, typename FloatN>
    struct cabxpyAx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      cabxpyAx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { x *= a.x; _caxpy(b, x, y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 5; } //! flops per element
    };

    void cabxpyAx(const double &a, const Complex &b,
		  ColorSpinorField &x, ColorSpinorField &y) {
      // swap arguments around
      blasCuda<cabxpyAx_,1,1,0,0>(make_double2(a,0.0), make_double2(REAL(b),IMAG(b)),
				  make_double2(0.0,0.0), x, y, x, x);
    }

    /**
       Functor performing the operation z[i] = a*x[i] + b*y[i] + z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypz_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpbypz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, z); _caxpy(b, y, z); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void caxpbypz(const Complex &a, ColorSpinorField &x, const Complex &b,
		  ColorSpinorField &y, ColorSpinorField &z) {
      blasCuda<caxpbypz_,0,0,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b),IMAG(b)),
				  make_double2(0.0,0.0), x, y, z, z);
    }

    /**
       Functor Performing the operation w[i] = a*x[i] + b*y[i] + c*z[i] + w[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypczpw_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      const Float2 c;
      caxpbypczpw_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b), c(c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, w); _caxpy(b, y, w); _caxpy(c, z, w); }

      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 12; } //! flops per element
    };

    void caxpbypczpw(const Complex &a, ColorSpinorField &x, const Complex &b,
		     ColorSpinorField &y, const Complex &c, ColorSpinorField &z,
		     ColorSpinorField &w) {
      blasCuda<caxpbypczpw_,0,0,0,1>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b),IMAG(b)),
				     make_double2(REAL(c),IMAG(c)), x, y, z, w);
    }

    /**
       double caxpyXmaz(c a, V x, V y, V z){}

       First performs the operation y[i] += a*x[i]
       Second performs the operator x[i] -= a*z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpyxmaz_ : public BlasFunctor<Float2,FloatN> {
      Float2 a;
      caxpyxmaz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); _caxpy(-a, z, x); }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void caxpyXmaz(const Complex &a, ColorSpinorField &x,
		   ColorSpinorField &y, ColorSpinorField &z) {
      blasCuda<caxpyxmaz_,1,1,0,0>(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0),
				   make_double2(0.0, 0.0), x, y, z, x);
    }

    /**
       double caxpyXmazMR(c a, V x, V y, V z){}

       First performs the operation y[i] += a*x[i]
       Second performs the operator x[i] -= a*z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpyxmazMR_ : public BlasFunctor<Float2,FloatN> {
      Float2 a;
      double3 *Ar3;
      caxpyxmazMR_(const Float2 &a, const Float2 &b, const Float2 &c)
	: a(a), Ar3(static_cast<double3*>(blas::getDeviceReduceBuffer())) { ; }

      inline __device__ __host__ void init() {
#ifdef __CUDA_ARCH__
	typedef decltype(a.x) real;
	double3 result = __ldg(Ar3);
	a.y = a.x * (real)(result.y) * ((real)1.0 / (real)result.z);
	a.x = a.x * (real)(result.x) * ((real)1.0 / (real)result.z);
#endif
      }

      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); _caxpy(-a, z, x); }

      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void caxpyXmazMR(const Complex &a, ColorSpinorField &x,
		     ColorSpinorField &y, ColorSpinorField &z) {
      if (!commAsyncReduction())
	errorQuda("This kernel requires asynchronous reductions to be set");
      if (x.Location() == QUDA_CPU_FIELD_LOCATION)
	errorQuda("This kernel cannot be run on CPU fields");

      blasCuda<caxpyxmazMR_,1,1,0,0>(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0),
				     make_double2(0.0, 0.0), x, y, z, x);
    }

    /**
       double tripleCGUpdate(d a, d b, V x, V y, V z, V w){}

       First performs the operation y[i] = y[i] + a*w[i]
       Second performs the operation z[i] = z[i] - a*x[i]
       Third performs the operation w[i] = z[i] + b*w[i]
    */
    template <typename Float2, typename FloatN>
    struct tripleCGUpdate_ : public BlasFunctor<Float2,FloatN> {
      Float2 a, b;
      tripleCGUpdate_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y += a.x*w; z -= a.x*x; w = z + b.x*w; }
      static int streams() { return 7; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    void tripleCGUpdate(const double &a, const double &b, ColorSpinorField &x,
			ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w) {
      if (x.Precision() != y.Precision()) {
      // call hacked mixed precision kernel
	mixed::blasCuda<tripleCGUpdate_,0,1,1,1>(make_double2(a,0.0), make_double2(b,0.0),
						 make_double2(0.0,0.0), x, y, z, w);
      } else {
	blasCuda<tripleCGUpdate_,0,1,1,1>(make_double2(a, 0.0), make_double2(b, 0.0),
					  make_double2(0.0, 0.0), x, y, z, w);
      }
    }

  } // namespace blas

} // namespace quda
