#include "hip/hip_runtime.h"
#include <transfer.h>
#include <gauge_field_order.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#if __COMPUTE_CAPABILITY__ >= 300
#include <generics/shfl.h>
#endif

namespace quda {

#ifdef GPU_MULTIGRID

  template <typename Float, typename F, typename G>
  struct DslashCoarseArg {
    F out;
    const F inA;
    const F inB;
    const G Y;
    const G X;
    const Float kappa;
    const int parity; // only use this for single parity fields
    const int nParity; // number of parities we're working on
    const int nFace;  // hard code to 1 for now
    const int dim[5];   // full lattice dimensions
    const int commDim[4]; // whether a given dimension is partitioned or not
    const int volumeCB;

    DslashCoarseArg(F &out, const F &inA, const F &inB, const G &Y, const G &X,
		    Float kappa, int parity, const ColorSpinorField &meta)
      : out(out), inA(inA), inB(inB), Y(Y), X(X), kappa(kappa), parity(parity),
	nParity(meta.SiteSubset()), nFace(1),
	dim{ (3-nParity) * meta.X(0), meta.X(1), meta.X(2), meta.X(3), meta.Ndim() == 5 ? meta.X(4) : 1 },
      commDim{comm_dim_partitioned(0), comm_dim_partitioned(1), comm_dim_partitioned(2), comm_dim_partitioned(3)},
      volumeCB(meta.VolumeCB()/dim[4])
    {  }
  };

  /**
     Applies the coarse dslash on a given parity and checkerboard site index

     @param out The result - kappa * Dslash in
     @param Y The coarse gauge field
     @param kappa Kappa value
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  extern __shared__ float s[];
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc, int color_stride, int dim_stride, int thread_dir, int thread_dim>
  __device__ __host__ inline void applyDslash(complex<Float> out[], DslashCoarseArg<Float,F,G> &arg, int x_cb, int src_idx, int parity, int s_row, int color_block, int color_offset) {
    const int their_spinor_parity = (arg.nParity == 2) ? (parity+1)&1 : 0;

    int coord[5];
    getCoords(coord, x_cb, arg.dim, parity);
    coord[4] = src_idx;

#ifdef __CUDA_ARCH__
    complex<Float> *shared_sum = (complex<Float>*)s;
    if (!thread_dir) {
#endif

      //Forward gather - compute fwd offset for spinor fetch
#pragma unroll
      for(int d = thread_dim; d < nDim; d+=dim_stride) // loop over dimension
      {
	const int fwd_idx = linkIndexP1(coord, arg.dim, d);

	if ( arg.commDim[d] && (coord[d] + arg.nFace >= arg.dim[d]) ) {
	  int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, arg.nFace);

#pragma unroll
	  for(int color_local = 0; color_local < Mc; color_local++) { //Color row
	    int c_row = color_block + color_local; // global color index
	    int row = s_row*Nc + c_row;
#pragma unroll
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
#pragma unroll
	      for(int c_col = 0; c_col < Nc; c_col+=color_stride) { //Color column
		int col = s_col*Nc + c_col + color_offset;
		out[color_local] += arg.Y(d+4, parity, x_cb, row, col)
		  * arg.inA.Ghost(d, 1, their_spinor_parity, ghost_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
	      }
	    }
	  }
	} else {
#pragma unroll
	  for(int color_local = 0; color_local < Mc; color_local++) { //Color row
	    int c_row = color_block + color_local; // global color index
	    int row = s_row*Nc + c_row;
#pragma unroll
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
#pragma unroll
	      for(int c_col = 0; c_col < Nc; c_col+=color_stride) { //Color column
		int col = s_col*Nc + c_col + color_offset;
		out[color_local] += arg.Y(d+4, parity, x_cb, row, col)
		  * arg.inA(their_spinor_parity, fwd_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
	      }
	    }
	  }
	}

      } // nDim

#if defined(__CUDA_ARCH__)
      if (thread_dim > 0) { // only need to write to shared memory if not master thread
#pragma unroll
	for (int color_local=0; color_local < Mc; color_local++) {
	  shared_sum[((color_local * blockDim.z + threadIdx.z )*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x] = out[color_local];
	}
      }
#endif

#ifdef __CUDA_ARCH__
    } else {
#endif

      //Backward gather - compute back offset for spinor and gauge fetch
#pragma unroll
      for(int d = thread_dim; d < nDim; d+=dim_stride)
	{
	const int back_idx = linkIndexM1(coord, arg.dim, d);
	const int gauge_idx = back_idx;
	if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	  const int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);
#pragma unroll
	  for (int color_local=0; color_local<Mc; color_local++) {
	    int c_row = color_block + color_local;
	    int row = s_row*Nc + c_row;
#pragma unroll
	    for (int s_col=0; s_col<Ns; s_col++)
#pragma unroll
	      for (int c_col=0; c_col<Nc; c_col+=color_stride) {
		int col = s_col*Nc + c_col + color_offset;
		out[color_local] += conj(arg.Y.Ghost(d, (parity+1)&1, ghost_idx, col, row))
		  * arg.inA.Ghost(d, 0, their_spinor_parity, ghost_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
	      }
	  }
	} else {
#pragma unroll
	  for(int color_local = 0; color_local < Mc; color_local++) {
	    int c_row = color_block + color_local;
	    int row = s_row*Nc + c_row;
#pragma unroll
	    for(int s_col = 0; s_col < Ns; s_col++)
#pragma unroll
	      for(int c_col = 0; c_col < Nc; c_col+=color_stride) {
		int col = s_col*Nc + c_col + color_offset;
		out[color_local] += conj(arg.Y(d, (parity+1)&1, gauge_idx, col, row))
		  * arg.inA(their_spinor_parity, back_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
	      }
	  }
	}

      } //nDim

#if defined(__CUDA_ARCH__)

#pragma unroll
      for (int color_local=0; color_local < Mc; color_local++) {
	shared_sum[ ((color_local * blockDim.z + threadIdx.z )*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x] = out[color_local];
      }

    } // forwards / backwards thread split
#endif

#ifdef __CUDA_ARCH__ // CUDA path has to recombine the foward and backward results
    __syncthreads();

    // (colorspin * dim_stride + dim * 2 + dir)
    if (thread_dim == 0 && thread_dir == 0) {

      // full split over dimension and direction
#pragma unroll
      for (int d=1; d<dim_stride; d++) { // get remaining forward fathers (if any)
	// 4-way 1,2,3  (stride = 4)
	// 2-way 1      (stride = 2)
#pragma unroll
	for (int color_local=0; color_local < Mc; color_local++) {
	  out[color_local] +=
	    shared_sum[(((color_local*blockDim.z/(2*dim_stride) + threadIdx.z/(2*dim_stride)) * 2 * dim_stride + d * 2 + 0)*blockDim.y+threadIdx.y)*blockDim.x+threadIdx.x];
	}
      }

#pragma unroll
      for (int d=0; d<dim_stride; d++) { // get all backward gathers
#pragma unroll
	for (int color_local=0; color_local < Mc; color_local++) {
	  out[color_local] +=
	    shared_sum[(((color_local*blockDim.z/(2*dim_stride) + threadIdx.z/(2*dim_stride)) * 2 * dim_stride + d * 2 + 1)*blockDim.y+threadIdx.y)*blockDim.x+threadIdx.x];
	}
      }

      // apply kappa
#pragma unroll
      for (int color_local=0; color_local<Mc; color_local++) out[color_local] *= -arg.kappa;

    }

#else // !__CUDA_ARCH__
    for (int color_local=0; color_local<Mc; color_local++) out[color_local] *= -arg.kappa;
#endif

    }

  /**
     Applies the coarse clover matrix on a given parity and
     checkerboard site index

     @param out The result out += X * in
     @param X The coarse clover field
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int Ns, int Nc, int Mc, int color_stride>
  __device__ __host__ inline void applyClover(complex<Float> out[], DslashCoarseArg<Float,F,G> &arg, int x_cb, int src_idx, int parity, int s, int color_block, int color_offset) {
    const int spinor_parity = (arg.nParity == 2) ? parity : 0;

    // M is number of colors per thread
#pragma unroll
    for(int color_local = 0; color_local < Mc; color_local++) {//Color out
      int c = color_block + color_local; // global color index
      int row = s*Nc + c;
#pragma unroll
      for(int s_col = 0; s_col < Ns; s_col++) //Spin in
#pragma unroll
	for(int c_col = 0; c_col < Nc; c_col+=color_stride) { //Color in
	  //Factor of kappa and diagonal addition now incorporated in X
	  int col = s_col*Nc + c_col + color_offset;
	  out[color_local] += arg.X(0, parity, x_cb, row, col) * arg.inB(spinor_parity, x_cb+src_idx*arg.volumeCB, s_col, c_col+color_offset);
	}
    }

  }

  //out(x) = M*in = \sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc, int color_stride,
	    int dim_thread_split, bool dslash, bool clover, int dir, int dim>
  __device__ __host__ inline void coarseDslash(DslashCoarseArg<Float,F,G> &arg, int x_cb, int src_idx, int parity, int s, int color_block, int color_offset)
  {
    complex <Float> out[Mc];
#pragma unroll
    for (int c=0; c<Mc; c++) out[c] = 0.0;
    if (dslash) applyDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dir,dim>(out, arg, x_cb, src_idx, parity, s, color_block, color_offset);
    if (clover && dir==0 && dim==0) applyClover<Float,F,G,Ns,Nc,Mc,color_stride>(out, arg, x_cb, src_idx, parity, s, color_block, color_offset);

    if (dir==0 && dim==0) {
      const int my_spinor_parity = (arg.nParity == 2) ? parity : 0;
#pragma unroll
      for (int color_local=0; color_local<Mc; color_local++) {
#if __CUDA_ARCH__ >= 300
	// reduce down to the first group of column-split threads
	const int warp_size = 32; // FIXME - this is buggy when x-dim * color_stride < 32
#pragma unroll
	for (int offset = warp_size/2; offset >= warp_size/color_stride; offset /= 2) out[color_local] += __shfl_down(out[color_local], offset);
#endif
	int c = color_block + color_local; // global color index
	if (color_offset == 0) arg.out(my_spinor_parity, x_cb+src_idx*arg.volumeCB, s, c) = out[color_local];
      }
    }
  }

  // CPU kernel for applying the coarse Dslash to a vector
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc, bool dslash, bool clover>
  void coarseDslash(DslashCoarseArg<Float,F,G> arg)
  {
    // the fine-grain parameters mean nothing for CPU variant
    const int color_stride = 1;
    const int color_offset = 0;
    const int dim_thread_split = 1;
    const int dir = 0;
    const int dim = 0;

    for (int parity= 0; parity < arg.nParity; parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int src_idx = 0; src_idx < arg.dim[4]; src_idx++) {
	//#pragma omp parallel for
	for(int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { // 4-d volume
	  for (int s=0; s<2; s++) {
	    for (int color_block=0; color_block<Nc; color_block+=Mc) { // Mc=Nc means all colors in a thread
	      coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dir,dim>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
	    }
	  }
	} // 4-d volumeCB
      } // src index
    } // parity

  }

  // GPU Kernel for applying the coarse Dslash to a vector
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc, int color_stride, int dim_thread_split, bool dslash, bool clover>
  __global__ void coarseDslashKernel(DslashCoarseArg<Float,F,G> arg)
  {
    constexpr int warp_size = 32;
    const int lane_id = threadIdx.x % warp_size;
    const int warp_id = threadIdx.x / warp_size;
    const int vector_site_width = warp_size / color_stride;

    int x_cb = blockIdx.x*(blockDim.x/color_stride) + warp_id*(warp_size/color_stride) + lane_id % vector_site_width;

    const int color_offset = lane_id / vector_site_width;

    // for full fields set parity from y thread index else use arg setting
    int paritySrc = blockDim.y*blockIdx.y + threadIdx.y;
    int src_idx = (arg.nParity == 2) ? paritySrc / 2 : paritySrc; // maybe want to swap order or source and parity for improved locality of same parity
    int parity = (arg.nParity == 2) ? paritySrc % 2 : arg.parity;

    // z thread dimension is (( s*(Nc/Mc) + color_block )*dim_thread_split + dim)*2 + dir
    int sMd = blockDim.z*blockIdx.z + threadIdx.z;
    int dir = sMd & 1;
    int sMdim = sMd >> 1;
    int dim = sMdim % dim_thread_split;
    int sM = sMdim / dim_thread_split;
    int s = sM / (Nc/Mc);
    int color_block = (sM % (Nc/Mc)) * Mc;

    if (x_cb >= arg.volumeCB) return;
    if (paritySrc >= arg.nParity * arg.dim[4]) return;

    if (dir == 0) {
      if (dim == 0)      coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,0,0>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 1) coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,0,1>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 2) coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,0,2>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 3) coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,0,3>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
    } else if (dir == 1) {
      if (dim == 0)      coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,1,0>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 1) coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,1,1>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 2) coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,1,2>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 3) coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,1,3>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
    }
  }

  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc, bool dslash, bool clover>
  class DslashCoarse : public Tunable {

  protected:
    DslashCoarseArg<Float,F,G> &arg;
    const ColorSpinorField &meta;

    const int max_color_col_stride = 4;
    mutable int color_col_stride;
    mutable int dim_threads;

    long long flops() const
    {
      return ((dslash*2*nDim+clover*1)*(8*Ns*Nc*Ns*Nc)-2*Ns*Nc)*arg.nParity*(long long)meta.VolumeCB();
    }
    long long bytes() const
    {
      return (dslash||clover) * arg.out.Bytes() + dslash*8*arg.inA.Bytes() + clover*arg.inB.Bytes() +
	arg.dim[4]*arg.nParity*(dslash*8*arg.Y.Bytes() + clover*arg.X.Bytes());
    }
    unsigned int sharedBytesPerThread() const { return (sizeof(complex<Float>) * Mc); }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions
    unsigned int minThreads() const { return color_col_stride * arg.volumeCB; } // 4-d volume since this x threads only
    unsigned int maxBlockSize() const { return deviceProp.maxThreadsPerBlock / (dim_threads * 2 * arg.nParity); }

    bool advanceBlockDim(TuneParam &param) const
    {
      dim3 block = param.block;
      dim3 grid = param.grid;
      bool ret = Tunable::advanceBlockDim(param);
      param.block.y = block.y; param.block.z = block.z;
      param.grid.y = grid.y; param.grid.z = grid.z;

      if (ret) { // we advanced the block.x so we're done
	return true;
      } else { // block.x (spacetime) was reset

	if (param.block.y < arg.nParity * arg.dim[4]) { // advance parity / 5th dimension
	  param.block.y++;
	  param.grid.y = (arg.nParity * arg.dim[4] + param.block.y - 1) / param.block.y;
	  return true;
	} else {
	  // reset parity / 5th dimension
	  param.block.y = 1;
	  param.grid.y = arg.nParity * arg.dim[4];

	  // let's try to advance spin/block-color
	  while(param.block.z <= dim_threads * 2 * 2 * (Nc/Mc)) {
	    param.block.z+=dim_threads * 2;
	    if ( (dim_threads*2*2*(Nc/Mc)) % param.block.z == 0) {
	      param.grid.z = (dim_threads * 2 * 2 * (Nc/Mc)) / param.block.z;
	      break;
	    }
	  }

	  // we can advance spin/block-color since this is valid
	  if (param.block.z <= dim_threads * 2 * 2 * (Nc/Mc) && param.block.z <= deviceProp.maxThreadsDim[2] ) { //
	    return true;
	  } else { // we have run off the end so let's reset
	    param.block.z = dim_threads * 2;
	    param.grid.z = 2 * (Nc/Mc);
	    return false;
	  }

	}
      }
    }

    int blockStep() const { return deviceProp.warpSize/4; }
    int blockMin() const { return deviceProp.warpSize/4; }

    // Experimental autotuning of the color column stride
    bool advanceAux(TuneParam &param) const
    {

#if __COMPUTE_CAPABILITY__ >= 300
      // we can only split the dot product on Kepler and later since we need the __shfl instruction
      if (2*param.aux.x <= max_color_col_stride && Nc % (2*param.aux.x) == 0 &&
	  param.block.x % deviceProp.warpSize == 0) {
	// An x-dimension block size that is not a multiple of the
	// warp size is incompatible with splitting the dot product
	// across the warp so we must skip this

	param.aux.x *= 2; // safe to advance
	color_col_stride = param.aux.x;

	// recompute grid size since minThreads() has now been updated
	param.grid.x = (minThreads()+param.block.x-1)/param.block.x;

	// check this grid size is valid before returning
	if (param.grid.x < deviceProp.maxGridSize[0]) return true;
      }
#endif

      // reset color column stride if too large or not divisible
      param.aux.x = 1;
      color_col_stride = param.aux.x;

      // recompute grid size since minThreads() has now been updated
      param.grid.x = (minThreads()+param.block.x-1)/param.block.x;

      if (2*param.aux.y <= nDim) {
	param.aux.y *= 2;
	dim_threads = param.aux.y;

	// need to reset z-block/grid size/shared_bytes since dim_threads has changed
	param.block.z = dim_threads * 2;
	param.grid.z = 2* (Nc / Mc);

	param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z > sharedBytesPerBlock(param) ?
	  sharedBytesPerThread()*param.block.x*param.block.y*param.block.z : sharedBytesPerBlock(param);

	return true;
      } else {
	param.aux.y = 1;
	dim_threads = param.aux.y;

	// need to reset z-block/grid size/shared_bytes since
	// dim_threads has changed.  Strictly speaking this isn't needed
	// since this is the outer dimension to tune, but would be
	// needed if we added an aux.z tuning dimension
	param.block.z = dim_threads * 2;
	param.grid.z = 2* (Nc / Mc);

	param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z > sharedBytesPerBlock(param) ?
	  sharedBytesPerThread()*param.block.x*param.block.y*param.block.z : sharedBytesPerBlock(param);

	return false;
      }
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      param.aux = make_int4(1,1,1,1);
      color_col_stride = param.aux.x;
      dim_threads = param.aux.y;

      Tunable::initTuneParam(param);
      param.block.y = 1;
      param.grid.y = arg.nParity * arg.dim[4];
      param.block.z = dim_threads * 2;
      param.grid.z = 2*(Nc/Mc);
      param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z > sharedBytesPerBlock(param) ?
	sharedBytesPerThread()*param.block.x*param.block.y*param.block.z : sharedBytesPerBlock(param);
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      param.aux = make_int4(1,1,1,1);
      color_col_stride = param.aux.x;
      dim_threads = param.aux.y;

      Tunable::defaultTuneParam(param);
      param.block.y = 1;
      param.grid.y = arg.nParity * arg.dim[4];
      param.block.z = dim_threads * 2;
      param.grid.z = 2*(Nc/Mc);
      param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z > sharedBytesPerBlock(param) ?
	sharedBytesPerThread()*param.block.x*param.block.y*param.block.z : sharedBytesPerBlock(param);
    }

  public:
    DslashCoarse(DslashCoarseArg<Float,F,G> &arg, const ColorSpinorField &meta)
      : arg(arg), meta(meta) {
      strcpy(aux, meta.AuxString());
#ifdef MULTI_GPU
      char comm[5];
      comm[0] = (arg.commDim[0] ? '1' : '0');
      comm[1] = (arg.commDim[1] ? '1' : '0');
      comm[2] = (arg.commDim[2] ? '1' : '0');
      comm[3] = (arg.commDim[3] ? '1' : '0');
      comm[4] = '\0';
      strcat(aux,",comm=");
      strcat(aux,comm);
#endif
    }
    virtual ~DslashCoarse() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	coarseDslash<Float,F,G,nDim,Ns,Nc,Mc,dslash,clover>(arg);
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

	switch (tp.aux.y) { // dimension gather parallelisation
	case 1:
	  switch (tp.aux.x) { // this is color_col_stride
	  case 1:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,1,1,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 2:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,2,1,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 4:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,4,1,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  default:
	    errorQuda("Color column stride %d not valid", tp.aux.x);
	  }
	  break;
	case 2:
	  switch (tp.aux.x) { // this is color_col_stride
	  case 1:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,1,2,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 2:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,2,2,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 4:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,4,2,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  default:
	    errorQuda("Color column stride %d not valid", tp.aux.x);
	  }
	  break;
	case 4:
	  switch (tp.aux.x) { // this is color_col_stride
	  case 1:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,1,4,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 2:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,2,4,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 4:
	    coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc,4,4,dslash,clover> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  default:
	    errorQuda("Color column stride %d not valid", tp.aux.x);
	  }
	  break;
	default:
	  errorQuda("Invalid dimension thread splitting %d", tp.aux.y);
	}
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(meta.VolString(), typeid(*this).name(), aux);
    }

  };


  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseColor,
	    int coarseSpin, QudaFieldLocation location>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash, bool clover) {
    typedef typename colorspinor::FieldOrderCB<Float,coarseSpin,coarseColor,1,csOrder> F;
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> G;

    F outAccessor(const_cast<ColorSpinorField&>(out));
    F inAccessorA(const_cast<ColorSpinorField&>(inA));
    F inAccessorB(const_cast<ColorSpinorField&>(inB));
    G yAccessor(const_cast<GaugeField&>(Y));
    G xAccessor(const_cast<GaugeField&>(X));
    DslashCoarseArg<Float,F,G> arg(outAccessor, inAccessorA, inAccessorB, yAccessor, xAccessor, (Float)kappa, parity, inA);

    const int colors_per_thread = 1;
    if (dslash) {
      if (clover) {
	DslashCoarse<Float,F,G,4,coarseSpin,coarseColor,colors_per_thread,true,true> dslash(arg, inA);
	dslash.apply(0);
      } else {
	DslashCoarse<Float,F,G,4,coarseSpin,coarseColor,colors_per_thread,true,false> dslash(arg, inA);
	dslash.apply(0);
      }
    } else {
      if (clover) {
	DslashCoarse<Float,F,G,4,coarseSpin,coarseColor,colors_per_thread,false,true> dslash(arg, inA);
	dslash.apply(0);
      } else {
	errorQuda("Unsupported dslash=false clover=false");
      }
    }
  }

  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseColor, int coarseSpin>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash, bool clover) {
    if (inA.Location() == QUDA_CUDA_FIELD_LOCATION) {
      ApplyCoarse<Float,csOrder,gOrder,coarseColor,coarseSpin,QUDA_CUDA_FIELD_LOCATION>
	(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    } else {
      ApplyCoarse<Float,csOrder,gOrder,coarseColor,coarseSpin,QUDA_CPU_FIELD_LOCATION>
	(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    }
  }

  // template on the number of coarse colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash, bool clover) {
    if (inA.Nspin() != 2)
      errorQuda("Unsupported number of coarse spins %d\n",inA.Nspin());

    if (inA.Ncolor() == 2) {
      ApplyCoarse<Float,csOrder,gOrder,2,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
#if 0
    } else if (inA.Ncolor() == 4) {
      ApplyCoarse<Float,csOrder,gOrder,4,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    } else if (inA.Ncolor() == 8) {
      ApplyCoarse<Float,csOrder,gOrder,8,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    } else if (inA.Ncolor() == 12) {
      ApplyCoarse<Float,csOrder,gOrder,12,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    } else if (inA.Ncolor() == 16) {
      ApplyCoarse<Float,csOrder,gOrder,16,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    } else if (inA.Ncolor() == 20) {
      ApplyCoarse<Float,csOrder,gOrder,20,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
#endif
    } else if (inA.Ncolor() == 24) {
      ApplyCoarse<Float,csOrder,gOrder,24,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
#if 0
    } else if (inA.Ncolor() == 28) {
      ApplyCoarse<Float,csOrder,gOrder,28,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
#endif
    } else if (inA.Ncolor() == 32) {
      ApplyCoarse<Float,csOrder,gOrder,32,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  template <typename Float>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash, bool clover) {

    if (Y.FieldOrder() != X.FieldOrder())
      errorQuda("Field order mismatch Y = %d, X = %d", Y.FieldOrder(), X.FieldOrder());

    if (inA.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch Y = %d, X = %d", Y.FieldOrder(), X.FieldOrder());

    if (inA.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER && Y.FieldOrder() == QUDA_FLOAT2_GAUGE_ORDER) {
      ApplyCoarse<Float,QUDA_FLOAT2_FIELD_ORDER, QUDA_FLOAT2_GAUGE_ORDER>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    } else if (inA.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER && Y.FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      ApplyCoarse<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER,QUDA_QDP_GAUGE_ORDER>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
    } else {
      errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", inA.FieldOrder(), Y.FieldOrder());
    }
  }

#endif // GPU_MULTIGRID

  struct DslashCoarseLaunch {

    ColorSpinorField &out;
    const ColorSpinorField &inA;
    const ColorSpinorField &inB;
    const GaugeField &Y;
    const GaugeField &X;
    double kappa;
    int parity;
    bool dslash;
    bool clover;

    DslashCoarseLaunch(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		       const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash, bool clover)
      : out(out), inA(inA), inB(inB), Y(Y), X(X), kappa(kappa), parity(parity), dslash(dslash), clover(clover) { }

    void operator()() {
#ifdef GPU_MULTIGRID
      if (inA.V() == out.V()) errorQuda("Aliasing pointers");

      if (out.Precision() != inA.Precision() || Y.Precision() != inA.Precision() || X.Precision() != inA.Precision())
	errorQuda("Precision mismatch out=%d inA=%d inB=%d Y=%d X=%d",
		  out.Precision(), inA.Precision(), inB.Precision(), Y.Precision(), X.Precision());

      // check all locations match
      Location(out, inA, inB, Y, X);

      inA.exchangeGhost((QudaParity)(1-parity), 0); // last parameter is dummy

      if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
	ApplyCoarse<double>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
#else
	errorQuda("Double precision multigrid has not been enabled");
#endif
      } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
	ApplyCoarse<float>(out, inA, inB, Y, X, kappa, parity, dslash, clover);
      } else {
	errorQuda("Unsupported precision %d\n", Y.Precision());
      }
#else
      errorQuda("Multigrid has not been built");
#endif
    }

  };

  // hooks into tune.cpp variables for policy tuning
  typedef std::map<TuneKey, TuneParam> map;
  const map& getTuneCache();

  void disableProfileCount();
  void enableProfileCount();

 class DslashCoarsePolicyTune : public Tunable {

   DslashCoarseLaunch &dslash;

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

 public:
   DslashCoarsePolicyTune(DslashCoarseLaunch &dslash) : dslash(dslash)
   {
      strcpy(aux,"policy,");
      if (dslash.dslash) strcat(aux,"dslash");
      strcat(aux, dslash.clover ? "clover," : ",");
      strcat(aux,dslash.inA.AuxString());
#ifdef MULTI_GPU
      char comm[5];
      comm[0] = (comm_dim_partitioned(0) ? '1' : '0');
      comm[1] = (comm_dim_partitioned(1) ? '1' : '0');
      comm[2] = (comm_dim_partitioned(2) ? '1' : '0');
      comm[3] = (comm_dim_partitioned(3) ? '1' : '0');
      comm[4] = '\0';
      strcat(aux,",comm=");
      strcat(aux,comm);
#endif

     // before we do policy tuning we must ensure the kernel
     // constituents have been tuned since we can't do nested tuning
     if (getTuneCache().find(tuneKey()) == getTuneCache().end()) {
       disableProfileCount();
       dslash();
       enableProfileCount();
     }
    }

   virtual ~DslashCoarsePolicyTune() { }

   void apply(const hipStream_t &stream) {
     TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
     dslash();
   }

   int tuningIter() const { return 10; }

   bool advanceTuneParam(TuneParam &param) const { return false; }

   TuneKey tuneKey() const {
     return TuneKey(dslash.inA.VolString(), typeid(*this).name(), aux);
   }

   long long flops() const {
     int nDim = 4;
     int Ns = dslash.inA.Nspin();
     int Nc = dslash.inA.Ncolor();
     int nParity = dslash.inA.SiteSubset();
     int volumeCB = dslash.inA.VolumeCB();
     return ((dslash.dslash*2*nDim+dslash.clover*1)*(8*Ns*Nc*Ns*Nc)-2*Ns*Nc)*nParity*volumeCB;
   }

   long long bytes() const {
     int nParity = dslash.inA.SiteSubset();
     return (dslash.dslash||dslash.clover) * dslash.out.Bytes() +
       dslash.dslash*8*dslash.inA.Bytes() + dslash.clover*dslash.inB.Bytes() +
       nParity*(dslash.dslash*dslash.Y.Bytes()*dslash.Y.VolumeCB()/(2*dslash.Y.Stride())
		+ dslash.clover*dslash.X.Bytes()/2);
     // multiply Y by volume / stride to correct for pad
   }
  };


  //Apply the coarse Dirac matrix to a coarse grid vector
  //out(x) = M*in = X*in - kappa*\sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  //Uses the kappa normalization for the Wilson operator.
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
	           const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash, bool clover) {

    DslashCoarseLaunch Dslash(out, inA, inB, Y, X, kappa, parity, dslash, clover);

    DslashCoarsePolicyTune policy(Dslash);
    policy.apply(0);

  }//ApplyCoarse


} // namespace quda
