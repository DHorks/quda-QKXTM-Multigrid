#include "hip/hip_runtime.h"
#include <tune_quda.h>
#include <clover_field_order.h>
#include <complex_quda.h>
#include <launch_kernel.cuh>
#include <face_quda.h>
#include <atomic.cuh>
#include <cub_helper.cuh>

namespace quda {

  using namespace clover;

#ifdef GPU_CLOVER_DIRAC

  template <typename Clover>
  struct CloverInvertArg : public ReduceArg<double2> {
    const Clover clover;
    Clover inverse;
    bool computeTraceLog;
//extra attributes for twisted mass clover
    bool twist;
    double mu2;
    CloverInvertArg(Clover &inverse, const Clover &clover, bool computeTraceLog=0) :
      ReduceArg<double2>(), inverse(inverse), clover(clover), computeTraceLog(computeTraceLog),
      twist(clover.Twisted()), mu2(clover.Mu2()) { }
  };

  /**
     Use a Cholesky decomposition to invert the clover matrix
     Here we use an inplace inversion which hopefully reduces register pressure
   */
  template <int blockSize, typename Float, typename Clover, bool computeTrLog, bool twist>
  __device__ __host__ inline double cloverInvertCompute(CloverInvertArg<Clover> arg, int x, int parity) {

    double trlogA = 0.0;

    for (int ch=0; ch<2; ch++) {
      Float A[36];
      // load the clover term into memory
      arg.clover.load(A, x, parity, ch);

      Float diag[6];
      Float tmp[6]; // temporary storage
      complex<Float> tri[15];

      // hack into the right order as MILC just to copy algorithm directly
      // FIXME use native ordering in the Cholseky 
      // factor of two is inherent to QUDA clover storage
      constexpr Float two = static_cast<Float>(2.0);
      for (int i=0; i<6; i++) diag[i] = two*A[i];

      const int idtab[15]={0,1,3,6,10,2,4,7,11,5,8,12,9,13,14};
#pragma unroll
      for (int i=0; i<15; i++) tri[idtab[i]] = complex<Float>(two*A[6+2*i], two*A[6+2*i+1]);

      //Compute (T^2 + mu2) first, then invert (not optimized!):
      if (twist) {
         complex<Float> aux[15];//hmmm, better to reuse A-regs...
         //another solution just to define (but compiler may not be happy with this, swapping everything in
         //the global buffer):
         //complex<Float>* aux = (complex<Float>*)&A[ch*36];
         //compute off-diagonal terms:
         aux[ 0] = tri[0]*diag[0]+diag[1]*tri[0]+conj(tri[2])*tri[1]+conj(tri[4])*tri[3]+conj(tri[7])*tri[6]+conj(tri[11])*tri[10];
         aux[ 1] = tri[1]*diag[0]+diag[2]*tri[1]+tri[2]*tri[0]+conj(tri[5])*tri[3]+conj(tri[8])*tri[6]+conj(tri[12])*tri[10];
         aux[ 2] = tri[2]*diag[1]+diag[2]*tri[2]+tri[1]*conj(tri[0])+conj(tri[5])*tri[4]+conj(tri[8])*tri[7]+conj(tri[12])*tri[11];
         aux[ 3] = tri[3]*diag[0]+diag[3]*tri[3]+tri[4]*tri[0]+tri[5]*tri[1]+conj(tri[9])*tri[6]+conj(tri[13])*tri[10];
         aux[ 4] = tri[4]*diag[1]+diag[3]*tri[4]+tri[3]*conj(tri[0])+tri[5]*tri[2]+conj(tri[9])*tri[7]+conj(tri[13])*tri[11];
         aux[ 5] = tri[5]*diag[2]+diag[3]*tri[5]+tri[3]*conj(tri[1])+tri[4]*conj(tri[2])+conj(tri[9])*tri[8]+conj(tri[13])*tri[12];
         aux[ 6] = tri[6]*diag[0]+diag[4]*tri[6]+tri[7]*tri[0]+tri[8]*tri[1]+tri[9]*tri[3]+conj(tri[14])*tri[10];
         aux[ 7] = tri[7]*diag[1]+diag[4]*tri[7]+tri[6]*conj(tri[0])+tri[8]*tri[2]+tri[9]*tri[4]+conj(tri[14])*tri[11];
         aux[ 8] = tri[8]*diag[2]+diag[4]*tri[8]+tri[6]*conj(tri[1])+tri[7]*conj(tri[2])+tri[9]*tri[5]+conj(tri[14])*tri[12];
         aux[ 9] = tri[9]*diag[3]+diag[4]*tri[9]+tri[6]*conj(tri[3])+tri[7]*conj(tri[4])+tri[8]*conj(tri[5])+conj(tri[14])*tri[13];
         aux[10] = tri[10]*diag[0]+diag[5]*tri[10]+tri[11]*tri[0]+tri[12]*tri[1]+tri[13]*tri[3]+tri[14]*tri[6];
         aux[11] = tri[11]*diag[1]+diag[5]*tri[11]+tri[10]*conj(tri[0])+tri[12]*tri[2]+tri[13]*tri[4]+tri[14]*tri[7];
         aux[12] = tri[12]*diag[2]+diag[5]*tri[12]+tri[10]*conj(tri[1])+tri[11]*conj(tri[2])+tri[13]*tri[5]+tri[14]*tri[8];
         aux[13] = tri[13]*diag[3]+diag[5]*tri[13]+tri[10]*conj(tri[3])+tri[11]*conj(tri[4])+tri[12]*conj(tri[5])+tri[14]*tri[9];
         aux[14] = tri[14]*diag[4]+diag[5]*tri[14]+tri[10]*conj(tri[6])+tri[11]*conj(tri[7])+tri[12]*conj(tri[8])+tri[13]*conj(tri[9]);

         //update diagonal elements:
         diag[0] = (Float)arg.mu2+diag[0]*diag[0]+norm(tri[ 0])+norm(tri[ 1])+norm(tri[ 3])+norm(tri[ 6])+norm(tri[10]);
         diag[1] = (Float)arg.mu2+diag[1]*diag[1]+norm(tri[ 0])+norm(tri[ 2])+norm(tri[ 4])+norm(tri[ 7])+norm(tri[11]); 
         diag[2] = (Float)arg.mu2+diag[2]*diag[2]+norm(tri[ 1])+norm(tri[ 2])+norm(tri[ 5])+norm(tri[ 8])+norm(tri[12]); 
         diag[3] = (Float)arg.mu2+diag[3]*diag[3]+norm(tri[ 3])+norm(tri[ 4])+norm(tri[ 5])+norm(tri[ 9])+norm(tri[13]); 
         diag[4] = (Float)arg.mu2+diag[4]*diag[4]+norm(tri[ 6])+norm(tri[ 7])+norm(tri[ 8])+norm(tri[ 9])+norm(tri[14]);
         diag[5] = (Float)arg.mu2+diag[5]*diag[5]+norm(tri[10])+norm(tri[11])+norm(tri[12])+norm(tri[13])+norm(tri[14]);

	 //update off-diagonal elements:
         for(int i = 0; i < 15; i++) tri[i] = aux[i];
      }

      for (int j=0; j<6; j++) {
	diag[j] = sqrt(diag[j]);
	tmp[j] = 1.0 / diag[j];

	for (int k=j+1; k<6; k++) {
	  int kj = k*(k-1)/2+j;
	  tri[kj] *= tmp[j];
	}

	for(int k=j+1;k<6;k++){
	  int kj=k*(k-1)/2+j;
	  diag[k] -= (tri[kj] * conj(tri[kj])).real();
	  for(int l=k+1;l<6;l++){
	    int lj=l*(l-1)/2+j;
	    int lk=l*(l-1)/2+k;
	    tri[lk] -= tri[lj] * conj(tri[kj]);
	  }
	}	
      }
      
      /* Accumulate trlogA */
      if (computeTrLog) for (int j=0;j<6;j++) trlogA += 2.0*log((double)(diag[j]));

      /* Now use forward and backward substitution to construct inverse */
      complex<Float> v1[6];
      for (int k=0;k<6;k++) {
	for(int l=0;l<k;l++) v1[l] = complex<Float>(0.0, 0.0);

	/* Forward substitute */
	v1[k] = complex<Float>(tmp[k], 0.0);
	for(int l=k+1;l<6;l++){
	  complex<Float> sum = complex<Float>(0.0, 0.0);
	  for(int j=k;j<l;j++){
	    int lj=l*(l-1)/2+j;		    
	    sum -= tri[lj] * v1[j];
	  }
	  v1[l] = sum * tmp[l];
	}
	
	/* Backward substitute */
	v1[5] = v1[5] * tmp[5];
	for(int l=4;l>=k;l--){
	  complex<Float> sum = v1[l];
	  for(int j=l+1;j<6;j++){
	    int jl=j*(j-1)/2+l;
	    sum -= conj(tri[jl]) * v1[j];
	  }
	  v1[l] = sum * tmp[l];
	}
	
	/* Overwrite column k */
	diag[k] = v1[k].real();
	for(int l=k+1;l<6;l++){
	  int lk=l*(l-1)/2+k;
	  tri[lk] = v1[l];
	}
      }

      constexpr Float half = static_cast<Float>(0.5);
      for (int i=0; i<6; i++) A[i] = half * diag[i];
#pragma unroll
      for (int i=0; i<15; i++) { A[6+2*i] = half*tri[idtab[i]].real(); A[6+2*i+1] = half*tri[idtab[i]].imag(); }

      // save the inverted matrix
      arg.inverse.save(A, x, parity, ch);
    }

    return trlogA;
  }

  template <int blockSize, typename Float, typename Clover, bool computeTrLog, bool twist>
  void cloverInvert(CloverInvertArg<Clover> arg) {  
    for (int parity=0; parity<2; parity++) {
      for (int x=0; x<arg.clover.volumeCB; x++) {
	// should make this thread safe if we ever apply threads to cpu code
	double trlogA = cloverInvertCompute<blockSize,Float,Clover,computeTrLog,twist>(arg, x, parity);
	if (computeTrLog) {
	  if (parity) arg.result_h[0].y += trlogA;
	  else arg.result_h[0].x += trlogA;
	}
      }
    }
  }

  template <int blockSize, typename Float, typename Clover, bool computeTrLog, bool twist>
  __launch_bounds__(2*blockSize)
  __global__ void cloverInvertKernel(CloverInvertArg<Clover> arg) {  
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = threadIdx.y;
    double trlogA_parity = 0.0;
    if (idx < arg.clover.volumeCB)
      trlogA_parity = cloverInvertCompute<blockSize,Float,Clover,computeTrLog,twist>(arg, idx, parity);
    double2 trlogA = parity ? make_double2(0.0,trlogA_parity) : make_double2(trlogA_parity, 0.0);

    if (computeTrLog) reduce2d<blockSize,2>(arg, trlogA);
  }

  template <typename Float, typename Clover>
  class CloverInvert : TunableLocalParity {
    CloverInvertArg<Clover> arg;
    const CloverField &meta; // used for meta data only
    const QudaFieldLocation location;

  private:
    bool tuneSharedBytes() const { return false; } // Don't tune the shared memory
    unsigned int minThreads() const { return arg.clover.volumeCB; }

  public:
    CloverInvert(CloverInvertArg<Clover> &arg, const CloverField &meta, QudaFieldLocation location) 
      : arg(arg), meta(meta), location(location) { 
      writeAuxString("stride=%d,prec=%lu,trlog=%s,twist=%s", arg.clover.stride, sizeof(Float),
		     arg.computeTraceLog ? "true" : "false", arg.twist ? "true" : "false");
    }
    virtual ~CloverInvert() { ; }
  
    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      arg.result_h[0] = make_double2(0.,0.);
      if (location == QUDA_CUDA_FIELD_LOCATION) {
	if (arg.computeTraceLog) {
	  if (arg.twist) {
	    LAUNCH_KERNEL_LOCAL_PARITY(cloverInvertKernel, tp, stream, arg, Float, Clover, true, true);
	  } else {
	    LAUNCH_KERNEL_LOCAL_PARITY(cloverInvertKernel, tp, stream, arg, Float, Clover, true, false);
	  }
	} else {
	  if (arg.twist) {
	    LAUNCH_KERNEL_LOCAL_PARITY(cloverInvertKernel, tp, stream, arg, Float, Clover, false, true);
	  } else {
	    LAUNCH_KERNEL_LOCAL_PARITY(cloverInvertKernel, tp, stream, arg, Float, Clover, false, false);
	  }
	}
      } else {
	if (arg.computeTraceLog) {
	  if (arg.twist) {
	    cloverInvert<1, Float, Clover, true, true>(arg);
	  } else {
	    cloverInvert<1, Float, Clover, true, false>(arg);
	  }
	} else {
	  if (arg.twist) {
	    cloverInvert<1, Float, Clover, false, true>(arg);
	  } else {
	    cloverInvert<1, Float, Clover, false, false>(arg);
	  }
	}
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(meta.VolString(), typeid(*this).name(), aux);
    }

    long long flops() const { return 0; } 
    long long bytes() const { return 2*arg.clover.volumeCB*(arg.inverse.Bytes() + arg.clover.Bytes()); } 

    void preTune() { if (arg.clover.clover == arg.inverse.clover) arg.inverse.save(); }
    void postTune() { if (arg.clover.clover == arg.inverse.clover) arg.inverse.load(); }

  };

  template <typename Float, typename Clover>
  void cloverInvert(Clover inverse, const Clover clover, bool computeTraceLog, 
		    double* const trlog, const CloverField &meta, QudaFieldLocation location) {
    CloverInvertArg<Clover> arg(inverse, clover, computeTraceLog);
    CloverInvert<Float,Clover> invert(arg, meta, location);
    invert.apply(0);

    if (arg.computeTraceLog) {
      hipDeviceSynchronize();
      comm_allreduce_array((double*)arg.result_h, 2);
      trlog[0] = arg.result_h[0].x;
      trlog[1] = arg.result_h[0].y;
    }
  }

  template <typename Float>
  void cloverInvert(const CloverField &clover, bool computeTraceLog, QudaFieldLocation location) {

    if (clover.isNative()) {
      typedef typename clover_mapper<Float>::type C;
      cloverInvert<Float>(C(clover, 1), C(clover, 0), computeTraceLog,
			  clover.TrLog(), clover, location);
    } else {
      errorQuda("Clover field %d order not supported", clover.Order());
    }

  }

#endif

  // this is the function that is actually called, from here on down we instantiate all required templates
  void cloverInvert(CloverField &clover, bool computeTraceLog, QudaFieldLocation location) {

#ifdef GPU_CLOVER_DIRAC
    if (clover.Precision() == QUDA_HALF_PRECISION && clover.Order() > 4) 
      errorQuda("Half precision not supported for order %d", clover.Order());

    if (clover.Precision() == QUDA_DOUBLE_PRECISION) {
      cloverInvert<double>(clover, computeTraceLog, location);
    } else if (clover.Precision() == QUDA_SINGLE_PRECISION) {
      cloverInvert<float>(clover, computeTraceLog, location);
    } else {
      errorQuda("Precision %d not supported", clover.Precision());
    }
#else
    errorQuda("Clover has not been built");
#endif
  }

} // namespace quda
