#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#include <tune_quda.h>

namespace quda {

  template <typename Field>
  struct PackGhostArg {

    Field field;
    void **ghost;
    const void *v;
    int X[QUDA_MAX_DIM];
    const int volumeCB;
    const int nDim;
    const int nFace;
    const int parity;
    const int nParity;
    const int dagger;
    const QudaDWFPCType pc_type;
    int commDim[4]; // whether a given dimension is partitioned or not

    PackGhostArg(Field field, void **ghost, const ColorSpinorField &a, int parity, int dagger)
      : field(field),
	ghost(ghost),
	v(a.V()),
	volumeCB(a.VolumeCB()),
	nDim(a.Ndim()),
	nFace(a.Nspin() == 1 ? 3 : 1),
	parity(parity),
	nParity(a.SiteSubset()),
	dagger(dagger),
	pc_type(a.DWFPCtype())
    {
      for (int d=0; d<nDim; d++) X[d] = a.X(d);
      X[0] *= (nParity == 1) ? 2 : 1; // set to full lattice dimensions
      X[4] = (nDim == 5) ? a.X(4) : 1; // set fifth dimension correctly
      for (int i=0; i<4; i++) {
	commDim[i] = comm_dim_partitioned(i);
      }
    }
  };

  template <typename Float, int Ns, int Nc, int Mc, typename Arg>
  __device__ __host__ inline void packGhost(Arg &arg, int cb_idx, int parity, int spinor_parity, int color_block) {
    typedef typename mapper<Float>::type RegType;

    const int *X = arg.X;
    int x[5] = { };
    if (arg.nDim == 5) getCoords5(x, cb_idx, X, parity, arg.pc_type);
    else getCoords(x, cb_idx, X, parity);

#pragma unroll
    for (int dim=0; dim<4; dim++) {
      if (arg.commDim[dim] && x[dim] < arg.nFace){
	for (int s=0; s<Ns; s++) {
	  for (int color_local=0; color_local<Mc; color_local++) {
	    int c = color_block + color_local;
	    arg.field.Ghost(dim, 0, spinor_parity, ghostFaceIndex<0>(x,arg.X,dim,arg.nFace), s, c)
	      = arg.field(spinor_parity, cb_idx, s, c);
	  }
	}
      }
      
      if (arg.commDim[dim] && x[dim] >= X[dim] - arg.nFace){
	for (int s=0; s<Ns; s++) {
	  for (int color_local=0; color_local<Mc; color_local++) {
	    int c = color_block + color_local;
	    arg.field.Ghost(dim, 1, spinor_parity, ghostFaceIndex<1>(x,arg.X,dim,arg.nFace), s, c)
	      = arg.field(spinor_parity, cb_idx, s, c);
	  }
	}
      }
    }
  }

  template <typename Float, int Ns, int Nc, int Mc, typename Arg>
  void GenericPackGhost(Arg &arg) {
    for (int parity=0; parity<arg.nParity; parity++) {
      parity = (arg.nParity == 2) ? parity : arg.parity;
      const int spinor_parity = (arg.nParity == 2) ? parity : 0;
      for (int i=0; i<arg.volumeCB; i++)
	for (int color_block=0; color_block<Nc; color_block+=Mc)
	  packGhost<Float,Ns,Nc,Mc>(arg, i, parity, spinor_parity, color_block);
    }
  }

  template <typename Float, int Ns, int Nc, int Mc, typename Arg>
  __global__ void GenericPackGhostKernel(Arg arg) {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    if (x_cb >= arg.volumeCB) return;
    const int parity = (arg.nParity == 2) ? blockDim.y*blockIdx.y + threadIdx.y : arg.parity;
    const int spinor_parity = (arg.nParity == 2) ? parity : 0;
    const int color_block = (blockDim.z*blockIdx.z + threadIdx.z)*Mc;
    if (color_block >= Nc) return;
    packGhost<Float,Ns,Nc,Mc>(arg, x_cb, parity, spinor_parity, color_block);
  }

  template <typename Float, int Ns, int Nc, int Mc, typename Arg>
  class GenericPackGhostLauncher : public Tunable {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const { return 0; }
    long long bytes() const {
      // FIXME take into account paritioning
      size_t totalBytes = 0;
      for (int d=0; d<4; d++) {
	if (!comm_dim_partitioned(d)) continue;
	totalBytes += 2*arg.nFace*2*Ns*Nc*meta.SurfaceCB(d)*meta.Precision();
      }
      return totalBytes;
    }

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

    bool advanceBlockDim(TuneParam &param) const
    {
      dim3 block = param.block;
      dim3 grid = param.grid;
      bool ret = Tunable::advanceBlockDim(param);
      param.block.y = block.y; param.block.z = block.z;
      param.grid.y = grid.y; param.grid.z = grid.z;

      if (ret) { // we advanced the block.x so we're done
	return true;
      } else { // block.x (spacetime) was reset

	// y thread dimension corresponds to parity
	// z thread dimension corresponds to block color (spin is kept thread local)
	if (param.block.y == 1 && arg.nParity == 2) { // advance parity
	  param.block.y = arg.nParity;
	  param.grid.y = 1;
	  return true;
	} else {
	  // reset parity
	  param.block.y = 1;
	  param.grid.y = arg.nParity;

	  // let's try to block-color
	  while(param.block.z <= Nc/Mc) {
	    param.block.z++;
	    if ( (Nc/Mc) % param.block.z == 0) {
	      param.grid.z = (Nc/Mc) / param.block.z;
	      break;
	    }
	  }

	  // we can advance block-color since this is valid
	  if (param.block.z <= Nc/Mc) { //
	    return true;
	  } else { // we have run off the end so let's reset
	    param.block.z = 1;
	    param.grid.z = Nc/Mc;
	    return false;
	  }

	}
      }
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.block.y = 1;
      param.grid.y = arg.nParity;
      param.block.z = 1;
      param.grid.z = Nc/Mc;
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.block.y = 1;
      param.grid.y = arg.nParity;
      param.block.z = 1;
      param.grid.z = Nc/Mc;
    }

  public:
    GenericPackGhostLauncher(Arg &arg, const ColorSpinorField &meta) : arg(arg), meta(meta) {
      strcpy(aux, meta.AuxString());
#ifdef MULTI_GPU
      char comm[5];
      comm[0] = (arg.commDim[0] ? '1' : '0');
      comm[1] = (arg.commDim[1] ? '1' : '0');
      comm[2] = (arg.commDim[2] ? '1' : '0');
      comm[3] = (arg.commDim[3] ? '1' : '0');
      comm[4] = '\0';
      strcat(aux,",comm=");
      strcat(aux,comm);
#endif
    }

    virtual ~GenericPackGhostLauncher() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	GenericPackGhost<Float,Ns,Nc,Mc,Arg>(arg);
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	GenericPackGhostKernel<Float,Ns,Nc,Mc,Arg> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString());
    }
  };

  template <typename Float, QudaFieldOrder order, int Ns, int Nc>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {

    typedef typename colorspinor::FieldOrderCB<Float,Ns,Nc,1,order> Q;
    Q field(a, 0, ghost);

    const int colors_per_thread = 1;
    PackGhostArg<Q> arg(field, ghost, a, parity, dagger);
    GenericPackGhostLauncher<Float,Ns,Nc,colors_per_thread,PackGhostArg<Q> > launch(arg, a);
    launch.apply(0);
  }

  template <typename Float, QudaFieldOrder order, int Ns>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {
    
    if (a.Ncolor() == 2) {
      genericPackGhost<Float,order,Ns,2>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 3) {
      genericPackGhost<Float,order,Ns,3>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 4) {
      genericPackGhost<Float,order,Ns,4>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 6) {
      genericPackGhost<Float,order,Ns,6>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 8) {
      genericPackGhost<Float,order,Ns,8>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 12) {
      genericPackGhost<Float,order,Ns,12>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 16) {
      genericPackGhost<Float,order,Ns,16>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 20) {
      genericPackGhost<Float,order,Ns,20>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 24) {
      genericPackGhost<Float,order,Ns,24>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 28) {
      genericPackGhost<Float,order,Ns,28>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 32) {
      genericPackGhost<Float,order,Ns,32>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 48) {
      genericPackGhost<Float,order,Ns,48>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 72) {
      genericPackGhost<Float,order,Ns,72>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 96) {
      genericPackGhost<Float,order,Ns,96>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 256) {
      genericPackGhost<Float,order,Ns,256>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 576) {
      genericPackGhost<Float,order,Ns,576>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 768) {
      genericPackGhost<Float,order,Ns,768>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 1024) {
      genericPackGhost<Float,order,Ns,1024>(ghost, a, parity, dagger);
    } else {
      errorQuda("Unsupported nColor = %d", a.Ncolor());
    }

  }

  template <typename Float, QudaFieldOrder order>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {

    if (a.Nspin() == 4) {
      genericPackGhost<Float,order,4>(ghost, a, parity, dagger);
    } else if (a.Nspin() == 2) {
      genericPackGhost<Float,order,2>(ghost, a, parity, dagger);
#ifdef GPU_STAGGERED_DIRAC
    } else if (a.Nspin() == 1) {
      genericPackGhost<Float,order,1>(ghost, a, parity, dagger);
#endif
    } else {
      errorQuda("Unsupported nSpin = %d", a.Nspin());
    }

  }

  template <typename Float>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {

    if (a.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      genericPackGhost<Float,QUDA_FLOAT2_FIELD_ORDER>(ghost, a, parity, dagger);
    } else if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      genericPackGhost<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(ghost, a, parity, dagger);
    } else {
      errorQuda("Unsupported field order = %d", a.FieldOrder());
    }

  }

  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {

    if (a.FieldOrder() == QUDA_QOP_DOMAIN_WALL_FIELD_ORDER) {
      errorQuda("Field order %d not supported", a.FieldOrder());
    }

    // only do packing if one of the dimensions is partitioned
    bool partitioned = false;
    for (int d=0; d<4; d++)
      if (comm_dim_partitioned(d)) partitioned = true;
    if (!partitioned) return;

    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      genericPackGhost<double>(ghost, a, parity, dagger);
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      genericPackGhost<float>(ghost, a, parity, dagger);
    } else {
      errorQuda("Unsupported precision %d", a.Precision());
    }

  }

} // namespace quda
