#include <gauge_field_order.h>
#include <copy_gauge_helper.cuh>

namespace quda {

  template <typename FloatOut, typename FloatIn, int length, typename InOrder>
  void copyGauge(const InOrder &inOrder, GaugeField &out, QudaFieldLocation location, 
		 FloatOut *Out, FloatOut **outGhost, int type) {
    int faceVolumeCB[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) faceVolumeCB[i] = out.SurfaceCB(i) * out.Nface(); 
    if (out.isNative()) {
      if (out.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	if (typeid(FloatOut)==typeid(short) && out.LinkType() == QUDA_ASQTAD_FAT_LINKS) {
	  copyGauge<short,FloatIn,length>
	    (FloatNOrder<short,length,2,19>(out, (short*)Out, (short**)outGhost), inOrder,
	     out.Volume(), faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
	} else {
	  typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_NO>::type G;
	  copyGauge<FloatOut,FloatIn,length>
	    (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	     out.Ndim(), out.Geometry(), out, location, type);
	}
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_12) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_12>::type G;
	copyGauge<FloatOut,FloatIn,length>
	  (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_8) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_8>::type G;
	copyGauge<FloatOut,FloatIn,length> 
	  (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
#ifdef GPU_STAGGERED_DIRAC
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_13) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_13>::type G;
        copyGauge<FloatOut,FloatIn,length>
	  (G(out, Out, outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(),  out.Geometry(), out, location, type);
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_9) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_9>::type G;
        copyGauge<FloatOut,FloatIn,length>
	  (G(out, Out, outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
#endif
      } else {
	errorQuda("Reconstruction %d and order %d not supported", out.Reconstruct(), out.Order());
      }
    } else if (out.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(QDPOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(), 
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(QDPJITOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(CPSOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(MILCOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(BQCDOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(TIFROrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", out.Order());
    }

  }

  template <typename FloatOut, typename FloatIn, int length>
    void copyGauge(GaugeField &out, const GaugeField &in, QudaFieldLocation location, 
		   FloatOut *Out, FloatIn *In, FloatOut **outGhost, FloatIn **inGhost, int type) {

    // reconstruction only supported on FloatN fields currently
    if (in.isNative()) {      
      if (in.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	if (typeid(FloatIn)==typeid(short) && in.LinkType() == QUDA_ASQTAD_FAT_LINKS) {
	  copyGauge<FloatOut,short,length> (FloatNOrder<short,length,2,19>
					    (in,(short*)In,(short**)inGhost),
					    out, location, Out, outGhost, type);
	} else {
	  typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_NO>::type G;
	  copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
	}
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_12) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_12>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_8) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_8>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
#ifdef GPU_STAGGERED_DIRAC
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_13) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_13>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_9) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_9>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
#endif
      } else {
	errorQuda("Reconstruction %d and order %d not supported", in.Reconstruct(), in.Order());
      }
    } else if (in.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(QDPOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(QDPJITOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(CPSOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(MILCOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(BQCDOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(TIFROrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", in.Order());
    }

  }

  void checkMomOrder(const GaugeField &u);

  template <typename FloatOut, typename FloatIn, int length, typename Out, typename In, typename Arg>
  void copyMom(Arg &arg, const GaugeField &meta, QudaFieldLocation location) {

    if (location == QUDA_CPU_FIELD_LOCATION) {
      copyGauge<FloatOut,FloatIn,length>(arg);
    } else if (location == QUDA_CUDA_FIELD_LOCATION) {
      CopyGauge<FloatOut,FloatIn,length, Out, In, 0> momCopier(arg, meta);
      momCopier.apply(0);
    } else {
      errorQuda("Undefined field location %d for copyMom", location);
    }

  }

  template <typename FloatOut, typename FloatIn>
  void copyGauge(GaugeField &out, const GaugeField &in, QudaFieldLocation location, FloatOut *Out, 
		 FloatIn *In, FloatOut **outGhost, FloatIn **inGhost, int type) {

    if (in.Ncolor() != 3 && out.Ncolor() != 3) {
      errorQuda("Unsupported number of colors; out.Nc=%d, in.Nc=%d", out.Ncolor(), in.Ncolor());
    }
    
    if (out.Geometry() != in.Geometry()) {
      errorQuda("Field geometries %d %d do not match", out.Geometry(), in.Geometry());
    }

    if (in.LinkType() != QUDA_ASQTAD_MOM_LINKS && out.LinkType() != QUDA_ASQTAD_MOM_LINKS) {
      // we are doing gauge field packing
      copyGauge<FloatOut,FloatIn,18>(out, in, location, Out, In, outGhost, inGhost, type);
    } else {
      if (out.Geometry() != QUDA_VECTOR_GEOMETRY) errorQuda("Unsupported geometry %d", out.Geometry());

      checkMomOrder(in);
      checkMomOrder(out);
    
      int faceVolumeCB[QUDA_MAX_DIM];
      for (int d=0; d<in.Ndim(); d++) faceVolumeCB[d] = in.SurfaceCB(d) * in.Nface();

      // momentum only currently supported on MILC (10), TIFR (18) and Float2 (10) fields currently
	if (out.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    typedef FloatNOrder<FloatOut,10,2,10> momOut;
	    typedef FloatNOrder<FloatIn,10,2,10> momIn;
	    CopyGaugeArg<momOut,momIn> arg(momOut(out, Out), momIn(in, In), in.Volume(),
				     faceVolumeCB, in.Ndim(), in.Geometry());
	    copyMom<FloatOut,FloatIn,10,momOut,momIn>(arg,in,location);
	  } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {
#ifdef BUILD_MILC_INTERFACE
	    typedef FloatNOrder<FloatOut,10,2,10> momOut;
	    typedef MILCOrder<FloatIn,10> momIn;
	    CopyGaugeArg<momOut,momIn> arg(momOut(out, Out), momIn(in, In), in.Volume(),
				     faceVolumeCB, in.Ndim(), in.Geometry());
	    copyMom<FloatOut,FloatIn,10,momOut,momIn>(arg,in,location);
#else
	    errorQuda("MILC interface has not been built\n");
#endif
	  } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {
#ifdef BUILD_TIFR_INTERFACE
	    typedef FloatNOrder<FloatOut,10,2,11> momOut;
	    typedef TIFROrder<FloatIn,18> momIn;
	    CopyGaugeArg<momOut,momIn> arg(momOut(out, Out), momIn(in, In), in.Volume(),
				     faceVolumeCB, in.Ndim(), in.Geometry());
	    copyMom<FloatOut,FloatIn,18,momOut,momIn>(arg,in,location);
#else
	    errorQuda("TIFR interface has not been built\n");
#endif
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
	} else if (out.Order() == QUDA_MILC_GAUGE_ORDER) {
	  typedef MILCOrder<FloatOut,10> momOut;
#ifdef BUILD_MILC_INTERFACE
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    typedef FloatNOrder<FloatIn,10,2,10> momIn;
	    CopyGaugeArg<momOut,momIn> arg(momOut(out, Out), momIn(in, In), in.Volume(),
				     faceVolumeCB, in.Ndim(), in.Geometry());
	    copyMom<FloatOut,FloatIn,10,momOut,momIn>(arg,in,location);
	  } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {
	    typedef MILCOrder<FloatIn,10> momIn;
	    CopyGaugeArg<momOut,momIn> arg(momOut(out, Out), momIn(in, In), in.Volume(),
				     faceVolumeCB, in.Ndim(), in.Geometry());
	    copyMom<FloatOut,FloatIn,10,momOut,momIn>(arg,in,location);
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
#else
	  errorQuda("MILC interface has not been built\n");
#endif
	} else if (out.Order() == QUDA_TIFR_GAUGE_ORDER) {
	  typedef TIFROrder<FloatOut,18> momOut;
#ifdef BUILD_TIFR_INTERFACE
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    // FIX ME - 11 is a misnomer to avoid confusion in template instantiation
	    typedef FloatNOrder<FloatIn,18,2,11> momIn;
	    CopyGaugeArg<momOut,momIn> arg(momOut(out, Out), momIn(in, In), in.Volume(),
				     faceVolumeCB, in.Ndim(), in.Geometry());
	    copyMom<FloatOut,FloatIn,18,momOut,momIn>(arg,in,location);
	  } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {
	    typedef TIFROrder<FloatIn,18> momIn;
	    CopyGaugeArg<momOut,momIn> arg(momOut(out, Out), momIn(in, In), in.Volume(),
				     faceVolumeCB, in.Ndim(), in.Geometry());
	    copyMom<FloatOut,FloatIn,18,momOut,momIn>(arg,in,location);
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
#else
	  errorQuda("TIFR interface has not been built\n");
#endif
	} else {
	  errorQuda("Gauge field orders %d not supported", out.Order());
	}
    }
  }


} // namespace quda
